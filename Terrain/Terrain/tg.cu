/************************************************************
*	ECE408 Parallel Programming - Final Project				*
*															*
*	Topic: Terrain Generation								*
*	Members: Lai,Haoming; Ma,Yunhan; Wang,Bangqi			*
*															*
************************************************************/

/* 
* Terrain Generation:
* Algorithmn: Diamond Square Algorithmn.
* Version:
*			0. Serial version: 1 * square loop + 4 * diamond loop;
*			1. Parallel version: 1 * sdsfsdfsdf + 4 * diamond kernel; 
*			2. Less Kernel Version: 1 * square kernal + 1 * simple diamond kernel (1 thread => 4 vertex);
*			3. Smarter Kernel Version: 1 * sqaure kernel + 1 * smart diamond kernel (1 thread => 1 vertex);
*			4. One Kernel Version: 1 * square_diamond kernel combined; (based on version 2)
*			5. Kernel Device Version: 1 * kernel + 1 * square device + 1 * diamond device;
*			6. Less Threads Version: 1 * kernel + 1 * square device + 1 * diamond device (only active threads we need);
*			7. Shared Memory Version: 1 * kernel + 1 * square device + 1 * diamond device (use share memory);
*
*			8. 2D Smarter Kernel Versio: 1 * sqaure kernel + 1 * smart diamond kernel (1 thread => 1 vertex);
*/

#include "hip/hip_runtime.h"

#include <hip/device_functions.h>
#include <stdio.h>

#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <math.h>

/* Set the parameter */
/* Choose the version to use */
//#define VERSION 0
/* Set the length of each edge. please put power of 2 */
#define SIZE 512
/* Set number of array */
#define N (SIZE+1)*(SIZE+1)
/* Set the roughness for terrain */
#define ROUGHNESS 10
/* Set the height for each corner */
#define CORNER 0

/* main function for different version */
int version_0 (void);
int version_1 (void);
int version_2 (void);
int version_3 (bool print, int block_size);
int version_4 (bool print);
int version_5 (void);
int version_6 (void);
int version_7 (void);
int version_8 (bool print, int block_size);
int version_9 (bool print, int block_size);

/* main function */
int main (void){
	int VERSION;
	int p;
	int block_size; 

	bool print= false;
	printf("what version do you want: ");
	scanf("%d", &VERSION);
	printf("print? (0/1): ");
	scanf("%d", &p);
	printf("please define block_size(max = 32): ");
	scanf("%d", &block_size);
	if(p)
		print = true; 

	switch(VERSION){
		/* test version 0 */
		case 0:
			version_0();
			break;
		case 1:
		/* test version 1 */
			version_1();
			break;
		case 2:
		/* test version 2 */
			version_2();
			break;
		case 3:
		/* test version 3 */
			version_3(print, block_size);
			break;
		case 4:
		/* test version 4 */
			version_4(print);
			break;
		case 5:
		/* test version 5 */
			version_5();
			break;
		case 6:
		/* test version 5 */
			version_6();
			break;
		case 7:
		/* test version 5 */
			version_7();
			break;
		case 8:
		/* test version 5 */
			version_8(print, block_size);
			break;
		case 9:
		/* test version 5 */
			version_9(print, block_size);
			break;
		default:
		/* test version 0 */
			version_0();
			return 0;
	}
}


///////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////// VERSION 0.0 ////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////
/*	VERSION 0.0: 
*			0.0 Serial version: 1 * square loop + 4 * loop;  
*			
*/

/* host code for serial version */
int version_0 (void) {
	clock_t start, end;
	double runTime;
	float heightMap[SIZE+1][SIZE+1];
	for(int i=0; i<SIZE; i++){
		for(int j=0; j<SIZE; j++){
			heightMap[i][j] = 0.0;
		}
	}

	//initial the first four points
	heightMap[0][0] = 0; 
	heightMap[SIZE][0] = 0; 
	heightMap[0][SIZE] = 0; 
	heightMap[SIZE][SIZE] = 0;  

	start = clock();
	int stride = SIZE;
	while(stride>=2){
		for(int i = 0; i<(SIZE/stride); i++){
			for(int j = 0; j<(SIZE/stride); j++){
				int leftbottom_x = i* stride;
				int leftbottom_y = j* stride;
				float average =  heightMap[leftbottom_x][leftbottom_y] + heightMap[leftbottom_x + stride][leftbottom_y] + heightMap[leftbottom_x][leftbottom_y+stride] + heightMap[leftbottom_x + stride][leftbottom_y +stride];
				average = average /4 ;
				heightMap[leftbottom_x + stride/2][leftbottom_y + stride/2]= average + rand() %10 - 5; 


				heightMap[leftbottom_x + stride/2 ][leftbottom_y] = (average + heightMap[leftbottom_x][leftbottom_y]  + heightMap[leftbottom_x + stride][leftbottom_y] ) /3 + rand() %10 -5; 
				heightMap[leftbottom_x][leftbottom_y + stride/2] = (average + heightMap[leftbottom_x][leftbottom_y]  + heightMap[leftbottom_x][leftbottom_y + stride] ) /3 + rand() %10 -5 ; 
				heightMap[leftbottom_x + stride][leftbottom_y+ stride/2] = (average + heightMap[leftbottom_x + stride ][leftbottom_y]  + heightMap[leftbottom_x + stride][leftbottom_y + stride] ) /3 +rand() %10-5; 
				heightMap[leftbottom_x+ stride/2][leftbottom_y+ stride] = (average + heightMap[leftbottom_x][leftbottom_y + stride]  + heightMap[leftbottom_x + stride][leftbottom_y + stride] ) /3 +rand() %10-5; 
			}


		}
		printf("%d \n", stride);
		stride = stride/2;
	}

 	for (int i=0; i<=SIZE; i++){
 		for(int j=0; j<=SIZE; j++){
	 		printf("%d: x = %d, y = %d; hm = %f\n", i*j, i, j, heightMap[i][j]);
		}
	}

	end = clock();
 	runTime = (double)(end - start)/CLOCKS_PER_SEC;

	printf("Run time for Version_0: %f\n", runTime);
	printf("Version 0\n");
	return 0;
}



///////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////// VERSION 1.0 ////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////
/*	VERSION 1.0: 
*			1.0 Parallel version: 1 * square kernel + 4 * diamond kernel;  
*			This parallel function parallelize the serial code directly. it change the one square loop to
*			one square kernel and change four diamond loop to four different diamond kernel.	1
*/

/* square kernel to calculate the middle point */
__global__ void Square_1(hiprandState* rng, float* hm, int rect, float* check1, float* check2){
	/* set idx */
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
  	if (idx < N){
		/* initialize vairable */
		int half = rect/2;
		int i, j, ni, nj, mi, mj;
		int squareInRow = SIZE/rect;
	  
		/* calculate vertex */
		i = (idx%squareInRow*rect)%SIZE;
		j = (idx/squareInRow*rect)%SIZE;
		ni = i + rect;
		nj = j + rect;
		mi = i + half;
		mj = j + half;

		/* set check value */
		check1[idx] = mi;
		check2[idx] = mj;

		/* set random generator */
		float v1 = (0.0f - (float)ROUGHNESS)/2;
		float v2 = ((float)ROUGHNESS)/2;
		hiprandState localState = rng[idx];
	    float rand = v1 + (v2 - v1) * hiprand_uniform(&localState);
		rng[idx] = localState;

	    /* set height map */
		hm[mi + mj*(SIZE+1)] = (hm[i + j*(SIZE+1)] + hm[ni + j*(SIZE+1)] + hm[i + nj*(SIZE+1)] + hm[ni + nj*(SIZE+1)])/4 + rand;
		__syncthreads();
  	}
}

/* diamond kernel 1_1 to calcualte middle bottom point */
__global__ void Diamond_1_1(hiprandState* rng, float* hm, int rect, float* check1, float* check2){
	/* set idx */
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
  	if (idx < N){
		/* initialize vairable */
		int half = rect/2;
		int i, mi, j;
		int pmi_b, pmj_b;
		float hm_b;
		int num_b;
		int squareInRow = SIZE/rect;
	  
		/* calculate vertex */
		i = (idx%squareInRow*rect)%SIZE;
		j = (idx/squareInRow*rect)%SIZE;
		mi = i + half;

		/* find 4 diamond vertex */
		/* bottom vertex */
		pmi_b = mi;
		pmj_b = j;

		/* set the value */
		/* bottom height */
		hm_b = 0;
		num_b = 3;
		if (pmj_b - half >= 0){
			hm_b += hm[pmi_b + (pmj_b-half)*(SIZE+1)];
			num_b = 4;
		}
		hm_b += hm[pmi_b + (pmj_b+half)*(SIZE+1)];
		hm_b += hm[(pmi_b-half) + pmj_b*(SIZE+1)];
		hm_b += hm[(pmi_b+half) + pmj_b*(SIZE+1)];

		/* set check value */
		// check1[idx] = hm_l;
		// check2[idx] = hm_l;

		/* set random generator */
		float v1 = (0.0f - (float)ROUGHNESS)/2;
		float v2 = ((float)ROUGHNESS)/2;
		hiprandState localState = rng[idx];
	    float rand1 = v1 + (v2 - v1) * hiprand_uniform(&localState);

	    /* set height map */
		hm[pmi_b + pmj_b*(SIZE+1)] = hm_b/num_b + rand1;
		// hm[mi + mj*(SIZE+1)] = (hm[i + j*(SIZE+1)] + hm[ni + j*(SIZE+1)] + hm[i + nj*(SIZE+1)] + hm[ni + nj*(SIZE+1)])/4 + rand;
		rng[idx] = localState;
		__syncthreads();     
  	}
  }

/* diamond kernel 1_2 to calcualte left point */
__global__ void Diamond_1_2(hiprandState* rng, float* hm, int rect, float* check1, float* check2){
	/* set idx */
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
  	if (idx < N){
		/* initialize vairable */
		int half = rect/2;
		int i, j, mj;
		int pmi_l, pmj_l;
		float hm_l;
		int num_l;
		int squareInRow = SIZE/rect;
	  
		/* calculate vertex */
		i = (idx%squareInRow*rect)%SIZE;
		j = (idx/squareInRow*rect)%SIZE;
		mj = j + half;

		/* find 4 diamond vertex */
		/* left vertex */
		pmi_l = i;
		pmj_l = mj;

		/* set the value */
		/* left height */
		hm_l = 0;
		num_l = 3;
		if (pmi_l - half >= 0){
			hm_l += hm[(pmi_l-half) + pmj_l*(SIZE+1)];
			num_l = 4;
		}
		hm_l += hm[(pmi_l+half) + pmj_l*(SIZE+1)];
		hm_l += hm[pmi_l + (pmj_l-half)*(SIZE+1)];
		hm_l += hm[pmi_l + (pmj_l+half)*(SIZE+1)];

		/* set check value */
		// check1[idx] = hm_l;
		// check2[idx] = hm_l;

		/* set random generator */
		float v1 = (0.0f - (float)ROUGHNESS)/2;
		float v2 = ((float)ROUGHNESS)/2;
		hiprandState localState = rng[idx];
	    float rand2 = v1 + (v2 - v1) * hiprand_uniform(&localState);

	    /* set height map */
      	hm[pmi_l + pmj_l*(SIZE+1)] = hm_l/num_l + rand2;

		// hm[mi + mj*(SIZE+1)] = (hm[i + j*(SIZE+1)] + hm[ni + j*(SIZE+1)] + hm[i + nj*(SIZE+1)] + hm[ni + nj*(SIZE+1)])/4 + rand;
		rng[idx] = localState;
		__syncthreads();     
  	}
}

/* diamond kernel 1_3 to calcualte right point */
__global__ void Diamond_1_3(hiprandState* rng, float* hm, int rect, float* check1, float* check2){
	/* set idx */
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
  	if (idx < N){
		/* initialize vairable */
		int half = rect/2;
		int i, j, ni, mj;
		int pmi_r, pmj_r;
		float hm_r;
		int num_r;
		int squareInRow = SIZE/rect;
	  
		/* calculate vertex */
		i = (idx%squareInRow*rect)%SIZE;
		j = (idx/squareInRow*rect)%SIZE;
		ni = i + rect;
		mj = j + half;

		/* find 4 diamond vertex */
		/* right vertex */
		pmi_r = ni;
		pmj_r = mj;

		/* set the value */
		/* right height */
		hm_r = 0;
		num_r = 3;
		if (pmi_r + half <= SIZE){
			hm_r += hm[(pmi_r+half) + pmj_r*(SIZE+1)];
			num_r = 4;
		}
		hm_r += hm[(pmi_r-half) + pmj_r*(SIZE+1)];
		hm_r += hm[pmi_r + (pmj_r-half)*(SIZE+1)];
		hm_r += hm[pmi_r + (pmj_r+half)*(SIZE+1)];

		/* set check value */
		// check1[idx] = hm_l;
		// check2[idx] = hm_l;

		/* get height for  */

		/* set random generator */
		float v1 = (0.0f - (float)ROUGHNESS)/2;
		float v2 = ((float)ROUGHNESS)/2;
		hiprandState localState = rng[idx];
	    float rand3 = v1 + (v2 - v1) * hiprand_uniform(&localState);

	    /* set height map */
      	hm[pmi_r + pmj_r*(SIZE+1)] = hm_r/num_r + rand3;
		// hm[mi + mj*(SIZE+1)] = (hm[i + j*(SIZE+1)] + hm[ni + j*(SIZE+1)] + hm[i + nj*(SIZE+1)] + hm[ni + nj*(SIZE+1)])/4 + rand;
		rng[idx] = localState;
		__syncthreads();     
  	}
}

/* diamond kernel 1_4 to calcualte middle top point */
__global__ void Diamond_1_4(hiprandState* rng, float* hm, int rect, float* check1, float* check2){
	/* set idx */
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
  	if (idx < N){
		/* initialize vairable */
		int half = rect/2;
		int i, j, mi, nj;
		int pmi_t, pmj_t;
		float hm_t;
		int num_t;
		int squareInRow = SIZE/rect;
	  
		/* calculate vertex */
		i = (idx%squareInRow*rect)%SIZE;
		j = (idx/squareInRow*rect)%SIZE;
		nj = j + rect;
		mi = i + half;

		/* find 4 diamond vertex */
		/* top vertex */
		pmi_t = mi;
		pmj_t = nj;

		/* set the value */
		/* top height */
		hm_t = 0;
		num_t = 3;
		if (pmj_t + half <= SIZE){
			hm_t += hm[pmi_t + (pmj_t+half)*(SIZE+1)];
			num_t = 4;
		}
		hm_t += hm[pmi_t + (pmj_t-half)*(SIZE+1)];
		hm_t += hm[(pmi_t-half) + pmj_t*(SIZE+1)];
		hm_t += hm[(pmi_t+half) + pmj_t*(SIZE+1)];

		/* set check value */
		// check1[idx] = hm_l;
		// check2[idx] = hm_l;

		/* get height for  */

		/* set random generator */
		float v1 = (0.0f - (float)ROUGHNESS)/2;
		float v2 = ((float)ROUGHNESS)/2;
		hiprandState localState = rng[idx];
	    float rand4 = v1 + (v2 - v1) * hiprand_uniform(&localState);

	    /* set height map */
      	hm[pmi_t + pmj_t*(SIZE+1)] = hm_t/num_t + rand4;  
		// hm[mi + mj*(SIZE+1)] = (hm[i + j*(SIZE+1)] + hm[ni + j*(SIZE+1)] + hm[i + nj*(SIZE+1)] + hm[ni + nj*(SIZE+1)])/4 + rand;
		rng[idx] = localState;
		__syncthreads();     
  	}
}

/* host code for version 1.0 */
int version_1 (void) {
	printf("Version 1: square kernel + 4 diamond kernel\n");
	/* initialize variables */
	float check1[N];
	float check2[N];
	float heightMap[N];
	/* initialize device */
	float *dev_heightMap;
	float *dev_check1;
	float *dev_check2;
	/* initialize time*/
	clock_t start, end;
	double runTime;
	/* initial height map */
	for (int i=0; i<N; i++){
	  heightMap[i] = 0;
	}

	/* set height for corner */
	heightMap[0 + 0 * (SIZE+1)] = CORNER;
	printf("heightMap_corner0: %f\n", heightMap[0 + 0 * (SIZE+1)]);
	heightMap[SIZE + 0 * (SIZE+1)] = CORNER;
	printf("heightMap_corner1: %f\n", heightMap[SIZE + 0 * (SIZE+1)]);
	heightMap[0 + SIZE * (SIZE+1)] = CORNER;
	printf("heightMap_corner3: %f\n", heightMap[0 + SIZE * (SIZE+1)]);
	heightMap[SIZE + SIZE * (SIZE+1)] = CORNER;
	printf("heightMap_corner2: %f\n", heightMap[SIZE + SIZE * (SIZE+1)]);

	hiprandState* rng;
	/* allocate memory for device */
	hipMalloc(&rng, N * sizeof(hiprandState));
	hipMalloc((void**)&dev_heightMap, N * sizeof(float));
	hipMalloc((void**)&dev_check1, N * sizeof(float));
	hipMalloc((void**)&dev_check2, N * sizeof(float));

	/* memory copy from host to device */
	hipMemcpy(dev_heightMap, heightMap, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_check1, check1, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_check2, check2, N * sizeof(float), hipMemcpyHostToDevice);

	start = clock();
	/* run kernel */
 	for (int i=SIZE; i>1; i=i/2){
		Square_1<<<ceil((float)N/256),256>>>(rng, (float*)dev_heightMap, i, dev_check1, dev_check2);
		hipDeviceSynchronize();
		Diamond_1_1<<<ceil((float)N/256),256>>>(rng, (float*)dev_heightMap, i, dev_check1, dev_check2);
		hipDeviceSynchronize();
		Diamond_1_2<<<ceil((float)N/256),256>>>(rng, (float*)dev_heightMap, i, dev_check1, dev_check2);
		hipDeviceSynchronize();
		Diamond_1_3<<<ceil((float)N/256),256>>>(rng, (float*)dev_heightMap, i, dev_check1, dev_check2);
		hipDeviceSynchronize();
		Diamond_1_4<<<ceil((float)N/256),256>>>(rng, (float*)dev_heightMap, i, dev_check1, dev_check2);
		hipDeviceSynchronize();
 	}
 	end = clock();

	/* memory copy from device to host*/
	hipMemcpy(heightMap, dev_heightMap, N * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(check1, dev_check1, N * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(check2, dev_check2, N * sizeof(float), hipMemcpyDeviceToHost);

	/* print the output */
	for (int i=0; i<N; i++){
	  printf("%d: x = %d, y = %d; hm = %f\n", i, i%(SIZE+1), i/(SIZE+1), heightMap[i]);
	}

	// printf("%f\n", cpu_time_used);
	hipFree(dev_heightMap);
	hipFree(dev_check1);
	hipFree(dev_check2);

 	runTime = (double)(end - start)/CLOCKS_PER_SEC;
	printf("Run time for Version_1: %f\n", runTime);
	return EXIT_SUCCESS;
}




///////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////// VERSION 2.0 ////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////
/*	VERSION 2.0:
*			2.0 Less Kernel Version: 1 * square kernal + 1 * simple diamond kernel (1 thread => 4 vertex); 
*			This kernel combine the four diamond kernel to one single kernel. However, each thread in diamond
*			kernel needs to calculate four vertex.
*/

/* combined diamond kernel to calculate the four point in each thread */
__global__ void Diamond_2(hiprandState* rng, float* hm, int rect, float* check1, float* check2){
	/* set idx */
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
  	if (idx < N){
		/* initialize vairable */
		int half = rect/2;
		int i, j, ni, nj, mi, mj;
		int pmi_b, pmj_b, pmi_l, pmj_l, pmi_r, pmj_r, pmi_t, pmj_t;
		float hm_b, hm_l, hm_r, hm_t;
		int num_b, num_l, num_r, num_t;
		int squareInRow = SIZE/rect;
	  
		/* calculate vertex */
		i = (idx%squareInRow*rect)%SIZE;
		j = (idx/squareInRow*rect)%SIZE;
		ni = i + rect;
		nj = j + rect;
		mi = i + half;
		mj = j + half;

		/* find 4 diamond vertex */
		/* bottom vertex */
		pmi_b = mi;
		pmj_b = j;
		/* left vertex */
		pmi_l = i;
		pmj_l = mj;
		/* right vertex */
		pmi_r = ni;
		pmj_r = mj;
		/* top vertex */
		pmi_t = mi;
		pmj_t = nj;

		/* set the value */
		/* bottom height */
		hm_b = 0;
		num_b = 3;
		if (pmj_b - half >= 0){
			hm_b += hm[pmi_b + (pmj_b-half)*(SIZE+1)];
			num_b = 4;
		}
		hm_b += hm[pmi_b + (pmj_b+half)*(SIZE+1)];
		hm_b += hm[(pmi_b-half) + pmj_b*(SIZE+1)];
		hm_b += hm[(pmi_b+half) + pmj_b*(SIZE+1)];

		/* left height */
		hm_l = 0;
		num_l = 3;
		if (pmi_l - half >= 0){
			hm_l += hm[(pmi_l-half) + pmj_l*(SIZE+1)];
			num_l = 4;
		}
		hm_l += hm[(pmi_l+half) + pmj_l*(SIZE+1)];
		hm_l += hm[pmi_l + (pmj_l-half)*(SIZE+1)];
		hm_l += hm[pmi_l + (pmj_l+half)*(SIZE+1)];

		/* right height */
		hm_r = 0;
		num_r = 3;
		if (pmi_r + half <= SIZE){
			hm_r += hm[(pmi_r+half) + pmj_r*(SIZE+1)];
			num_r = 4;
		}
		hm_r += hm[(pmi_r-half) + pmj_r*(SIZE+1)];
		hm_r += hm[pmi_r + (pmj_r-half)*(SIZE+1)];
		hm_r += hm[pmi_r + (pmj_r+half)*(SIZE+1)];

		/* top height */
		hm_t = 0;
		num_t = 3;
		if (pmj_t + half <= SIZE){
			hm_t += hm[pmi_t + (pmj_t+half)*(SIZE+1)];
			num_t = 4;
		}
		hm_t += hm[pmi_t + (pmj_t-half)*(SIZE+1)];
		hm_t += hm[(pmi_t-half) + pmj_t*(SIZE+1)];
		hm_t += hm[(pmi_t+half) + pmj_t*(SIZE+1)];

		/* set check value */
		check1[idx] = hm_l;
		check2[idx] = hm_l;

		/* get height for  */

		/* set random generator */
		float v1 = (0.0f - (float)ROUGHNESS)/2;
		float v2 = ((float)ROUGHNESS)/2;
		hiprandState localState = rng[idx];
	    float rand1 = v1 + (v2 - v1) * hiprand_uniform(&localState);
	    float rand2 = v1 + (v2 - v1) * hiprand_uniform(&localState);
	    float rand3 = v1 + (v2 - v1) * hiprand_uniform(&localState);
	    float rand4 = v1 + (v2 - v1) * hiprand_uniform(&localState);

	    /* set height map */
		hm[pmi_b + pmj_b*(SIZE+1)] = hm_b/num_b + rand1;
      	hm[pmi_l + pmj_l*(SIZE+1)] = hm_l/num_l + rand2;
      	hm[pmi_r + pmj_r*(SIZE+1)] = hm_r/num_r + rand3;
      	hm[pmi_t + pmj_t*(SIZE+1)] = hm_t/num_t + rand4;  
		// hm[mi + mj*(SIZE+1)] = (hm[i + j*(SIZE+1)] + hm[ni + j*(SIZE+1)] + hm[i + nj*(SIZE+1)] + hm[ni + nj*(SIZE+1)])/4 + rand;
		rng[idx] = localState;
		__syncthreads();     
  	}
}


/* the host code for version 2: 1 square kernel + 1 stupid diamond kernel */
int version_2 (void) {
	printf("Version 2: square kernel + stupid diamond kernel\n");
	/* initialize variables */
	float check1[N];
	float check2[N];
	float heightMap[N];
	/* initialize device */
	float *dev_heightMap;
	float *dev_check1;
	float *dev_check2;
	/* initialize time*/
	clock_t start, end;
	double runTime;
	/* initial height map */
	for (int i=0; i<N; i++){
	  heightMap[i] = 0;
	}

	/* set height for corner */
	heightMap[0 + 0 * (SIZE+1)] = CORNER;
	printf("heightMap_corner0: %f\n", heightMap[0 + 0 * (SIZE+1)]);
	heightMap[SIZE + 0 * (SIZE+1)] = CORNER;
	printf("heightMap_corner1: %f\n", heightMap[SIZE + 0 * (SIZE+1)]);
	heightMap[0 + SIZE * (SIZE+1)] = CORNER;
	printf("heightMap_corner3: %f\n", heightMap[0 + SIZE * (SIZE+1)]);
	heightMap[SIZE + SIZE * (SIZE+1)] = CORNER;
	printf("heightMap_corner2: %f\n", heightMap[SIZE + SIZE * (SIZE+1)]);

	hiprandState* rng;
	/* allocate memory for device */
	hipMalloc(&rng, N * sizeof(hiprandState));
	hipMalloc((void**)&dev_heightMap, N * sizeof(float));
	hipMalloc((void**)&dev_check1, N * sizeof(float));
	hipMalloc((void**)&dev_check2, N * sizeof(float));

	/* memory copy from host to device */
	hipMemcpy(dev_heightMap, heightMap, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_check1, check1, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_check2, check2, N * sizeof(float), hipMemcpyHostToDevice);

	start = clock();
	/* run kernel */
 	for (int i=SIZE; i>1; i=i/2){
		Square_1<<<ceil((float)N/256),256>>>(rng, (float*)dev_heightMap, i, dev_check1, dev_check2);
		hipDeviceSynchronize();
		Diamond_2<<<ceil((float)N/256),256>>>(rng, (float*)dev_heightMap, i, dev_check1, dev_check2);
		hipDeviceSynchronize();
 	}
 	end = clock();

	/* memory copy from device to host*/
	hipMemcpy(heightMap, dev_heightMap, N * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(check1, dev_check1, N * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(check2, dev_check2, N * sizeof(float), hipMemcpyDeviceToHost);

	/* print the output */
	for (int i=0; i<N; i++){
	  printf("%d: x = %d, y = %d; hm = %f\n", i, i%(SIZE+1), i/(SIZE+1), heightMap[i]);
	}

	// printf("%f\n", cpu_time_used);
	hipFree(dev_heightMap);
	hipFree(dev_check1);
	hipFree(dev_check2);

 	runTime = (double)(end - start)/CLOCKS_PER_SEC;
	printf("Run time for Version_2: %0.20f\n", runTime);
	return EXIT_SUCCESS;
}




///////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////// VERSION 3.0 ////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////
/*	VERSION 3.0:
*			3.0 Smarter Kernel Version: 1 * sqaure kernel + 1 * smart diamond kernel (1 thread => 1 vertex);
*			This version reconstruct the diamond kernel to use different threads for different vertx. Each 
*			thread in diamond kernel only need to calculate one vertex.
*/

/* smart diamond kernel calculate the diamond vertex and each thread only calculate one vertex */
__global__ void Diamond_3(hiprandState* rng, float* hm, int rect, float* check1, float* check2){
	/* set idx */
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
  	if (idx < N){
		/* initialize vairable */
		int half = rect/2;
		int i, j;
		int pmi, pmj;
		float hm_p;
		int num_p;
		int squareInRow = SIZE/rect;
	  
		/* calculate vertex */
		i = (idx%squareInRow*rect)%SIZE;
		j = (idx/squareInRow*rect)%SIZE;

		/* Calculate the diamond vertex use idx */
		int tid = idx/(squareInRow*squareInRow)%4;
		pmi = i + (1 - tid%2)*half + tid/2*half;
		pmj = j + tid%2*half + tid/2*half;

		/* Set the value */
		hm_p = 0;
		num_p = 0;
		if (pmi - half >= 0){
			hm_p += hm[(pmi-half) + pmj*(SIZE+1)];
			num_p++;
		}
		if (pmi + half <= SIZE){
			hm_p += hm[(pmi+half) + pmj*(SIZE+1)];
			num_p++;
		}
		if (pmj - half >= 0){
			hm_p += hm[pmi + (pmj-half)*(SIZE+1)];
			num_p++;
		}
		if (pmj + half <= SIZE){
			hm_p += hm[pmi + (pmj+half)*(SIZE+1)];
			num_p++;
		}

		/* set check value */
		check1[idx] = pmi;
		check2[idx] = pmj;

		/* set random generator */
		float v1 = (0.0f - (float)ROUGHNESS)/2;
		float v2 = ((float)ROUGHNESS)/2;
		hiprandState localState = rng[idx];
	    float rand = v1 + (v2 - v1) * hiprand_uniform(&localState);

		/* get height for  */
		hm[pmi + pmj*(SIZE+1)] = hm_p/num_p + rand;
		rng[idx] = localState;
		__syncthreads();    
  	}
}

/* the host code for version 3: 1 square kernel + 1 smart diamond kernel */
int version_3 (bool print, int block_size) {
	printf("Version 3: square kernel + smart diamond kernel\n");
	/* initialize variables */
	float check1[N];
	float check2[N];
	float heightMap[N];
	/* initialize device */
	float *dev_heightMap;
	float *dev_check1;
	float *dev_check2;
	/* initialize time*/
	clock_t start, end;
	double runTime;
	int size = block_size * block_size;
	/* initial height map */
	for (int i=0; i<N; i++){
	  heightMap[i] = 0;
	}

	/* set height for corner */
	heightMap[0 + 0 * (SIZE+1)] = CORNER;
	printf("heightMap_corner0: %f\n", heightMap[0 + 0 * (SIZE+1)]);
	heightMap[SIZE + 0 * (SIZE+1)] = CORNER;
	printf("heightMap_corner1: %f\n", heightMap[SIZE + 0 * (SIZE+1)]);
	heightMap[0 + SIZE * (SIZE+1)] = CORNER;
	printf("heightMap_corner3: %f\n", heightMap[0 + SIZE * (SIZE+1)]);
	heightMap[SIZE + SIZE * (SIZE+1)] = CORNER;
	printf("heightMap_corner2: %f\n", heightMap[SIZE + SIZE * (SIZE+1)]);

	hiprandState* rng;
	/* allocate memory for device */
	hipMalloc(&rng, N * sizeof(hiprandState));
	hipMalloc((void**)&dev_heightMap, N * sizeof(float));
	hipMalloc((void**)&dev_check1, N * sizeof(float));
	hipMalloc((void**)&dev_check2, N * sizeof(float));

	/* memory copy from host to device */
	hipMemcpy(dev_heightMap, heightMap, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_check1, check1, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_check2, check2, N * sizeof(float), hipMemcpyHostToDevice);

	start = clock();
	/* run kernel */
 	for (int i=SIZE; i>1; i=i/2){
		Square_1<<<ceil((float)N/size),size>>>(rng, (float*)dev_heightMap, i, dev_check1, dev_check2);
		hipDeviceSynchronize();
		Diamond_3<<<ceil((float)N/size),size>>>(rng, (float*)dev_heightMap, i, dev_check1, dev_check2);
		hipDeviceSynchronize();
 	}
 	end = clock();

	/* memory copy from device to host*/
	hipMemcpy(heightMap, dev_heightMap, N * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(check1, dev_check1, N * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(check2, dev_check2, N * sizeof(float), hipMemcpyDeviceToHost);

	/* print the output */
	if(print){
		for (int i=0; i<N; i++){
		  printf("%d: x = %d, y = %d; hm = %f\n", i, i%(SIZE+1), i/(SIZE+1), heightMap[i]);
		}
	}
	// printf("\n");	
	// for (int i=0; i<N; i++){
	//   printf("%d: pmi = %f, pmj = %f\n", i, check1[i], check2[i]);
	// }

	// printf("%f\n", cpu_time_used);
	hipFree(dev_heightMap);
	hipFree(dev_check1);
	hipFree(dev_check2);

 	runTime = (double)(end - start)/CLOCKS_PER_SEC;
	printf("Run time for Version_3: %0.20f\n", runTime);
	return EXIT_SUCCESS;
}


///////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////// VERSION 4.0 ////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////
/*	VERSION 4.0:
*			4.0 Less Kernel Version: 1 * square kernal + 1 * simple diamond kernel (1 thread => 4 vertex); 
*			This kernel combine the four diamond kernel to one single kernel. However, each thread in diamond
*			kernel needs to calculate four vertex.
*/
/* combined diamond kernel to calculate the four point in each thread */
__global__ void Square_Diamond_4(hiprandState* rng, float* hm, int rect, float* check1, float* check2){
	/* set idx */
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
  	if (idx < N){
  		/* initialize vairable */
		int half = rect/2;
		int i, j, ni, nj, mi, mj;
		int pmi_b, pmj_b, pmi_l, pmj_l, pmi_r, pmj_r, pmi_t, pmj_t;
		float hm_b, hm_l, hm_r, hm_t;
		int num_b, num_l, num_r, num_t;
		int squareInRow = SIZE/rect;

  		/* calculate vertex */
		i = (idx%squareInRow*rect)%SIZE;
		j = (idx/squareInRow*rect)%SIZE;
		ni = i + rect;
		nj = j + rect;
		mi = i + half;
		mj = j + half;

		/* set check value */
		check1[idx] = mi;
		check2[idx] = mj;

		/* set random generator */
		float v1 = (0.0f - (float)ROUGHNESS)/2;
		float v2 = ((float)ROUGHNESS)/2;
		hiprandState localState = rng[idx];
	    float rand = v1 + (v2 - v1) * hiprand_uniform(&localState);
		rng[idx] = localState;

	    /* set height map */
		hm[mi + mj*(SIZE+1)] = (hm[i + j*(SIZE+1)] + hm[ni + j*(SIZE+1)] + hm[i + nj*(SIZE+1)] + hm[ni + nj*(SIZE+1)])/4 + rand;
		__syncthreads();

		/* find 4 diamond vertex */
		/* bottom vertex */
		pmi_b = mi;
		pmj_b = j;
		/* left vertex */
		pmi_l = i;
		pmj_l = mj;
		/* right vertex */
		pmi_r = ni;
		pmj_r = mj;
		/* top vertex */
		pmi_t = mi;
		pmj_t = nj;

		/* set the value */
		/* bottom height */
		hm_b = 0;
		num_b = 3;
		if (pmj_b - half >= 0){
			hm_b += hm[pmi_b + (pmj_b-half)*(SIZE+1)];
			num_b = 4;
		}
		hm_b += hm[pmi_b + (pmj_b+half)*(SIZE+1)];
		hm_b += hm[(pmi_b-half) + pmj_b*(SIZE+1)];
		hm_b += hm[(pmi_b+half) + pmj_b*(SIZE+1)];

		/* left height */
		hm_l = 0;
		num_l = 3;
		if (pmi_l - half >= 0){
			hm_l += hm[(pmi_l-half) + pmj_l*(SIZE+1)];
			num_l = 4;
		}
		hm_l += hm[(pmi_l+half) + pmj_l*(SIZE+1)];
		hm_l += hm[pmi_l + (pmj_l-half)*(SIZE+1)];
		hm_l += hm[pmi_l + (pmj_l+half)*(SIZE+1)];

		/* right height */
		hm_r = 0;
		num_r = 3;
		if (pmi_r + half <= SIZE){
			hm_r += hm[(pmi_r+half) + pmj_r*(SIZE+1)];
			num_r = 4;
		}
		hm_r += hm[(pmi_r-half) + pmj_r*(SIZE+1)];
		hm_r += hm[pmi_r + (pmj_r-half)*(SIZE+1)];
		hm_r += hm[pmi_r + (pmj_r+half)*(SIZE+1)];

		/* top height */
		hm_t = 0;
		num_t = 3;
		if (pmj_t + half <= SIZE){
			hm_t += hm[pmi_t + (pmj_t+half)*(SIZE+1)];
			num_t = 4;
		}
		hm_t += hm[pmi_t + (pmj_t-half)*(SIZE+1)];
		hm_t += hm[(pmi_t-half) + pmj_t*(SIZE+1)];
		hm_t += hm[(pmi_t+half) + pmj_t*(SIZE+1)];

		/* set check value */
		check1[idx] = hm_l;
		check2[idx] = hm_l;

		/* get height for  */

		/* set random generator */
	    float rand1 = v1 + (v2 - v1) * hiprand_uniform(&localState);
	    float rand2 = v1 + (v2 - v1) * hiprand_uniform(&localState);
	    float rand3 = v1 + (v2 - v1) * hiprand_uniform(&localState);
	    float rand4 = v1 + (v2 - v1) * hiprand_uniform(&localState);

	    /* set height map */
		hm[pmi_b + pmj_b*(SIZE+1)] = hm_b/num_b + rand1;
      	hm[pmi_l + pmj_l*(SIZE+1)] = hm_l/num_l + rand2;
      	hm[pmi_r + pmj_r*(SIZE+1)] = hm_r/num_r + rand3;
      	hm[pmi_t + pmj_t*(SIZE+1)] = hm_t/num_t + rand4;  
		// hm[mi + mj*(SIZE+1)] = (hm[i + j*(SIZE+1)] + hm[ni + j*(SIZE+1)] + hm[i + nj*(SIZE+1)] + hm[ni + nj*(SIZE+1)])/4 + rand;
		rng[idx] = localState;
		__syncthreads();     
  	}
}


/* the host code for version 2: 1 square kernel + 1 stupid diamond kernel */
int version_4 (bool print) {
	printf("Version 2: square kernel + stupid diamond kernel\n");
	/* initialize variables */
	float check1[N];
	float check2[N];
	float heightMap[N];
	/* initialize device */
	float *dev_heightMap;
	float *dev_check1;
	float *dev_check2;
	/* initialize time*/
	clock_t start, end;
	double runTime;
	/* initial height map */
	for (int i=0; i<N; i++){
	  heightMap[i] = 0;
	}

	/* set height for corner */
	heightMap[0 + 0 * (SIZE+1)] = CORNER;
	printf("heightMap_corner0: %f\n", heightMap[0 + 0 * (SIZE+1)]);
	heightMap[SIZE + 0 * (SIZE+1)] = CORNER;
	printf("heightMap_corner1: %f\n", heightMap[SIZE + 0 * (SIZE+1)]);
	heightMap[0 + SIZE * (SIZE+1)] = CORNER;
	printf("heightMap_corner3: %f\n", heightMap[0 + SIZE * (SIZE+1)]);
	heightMap[SIZE + SIZE * (SIZE+1)] = CORNER;
	printf("heightMap_corner2: %f\n", heightMap[SIZE + SIZE * (SIZE+1)]);

	hiprandState* rng;
	/* allocate memory for device */
	hipMalloc(&rng, N * sizeof(hiprandState));
	hipMalloc((void**)&dev_heightMap, N * sizeof(float));
	hipMalloc((void**)&dev_check1, N * sizeof(float));
	hipMalloc((void**)&dev_check2, N * sizeof(float));

	/* memory copy from host to device */
	hipMemcpy(dev_heightMap, heightMap, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_check1, check1, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_check2, check2, N * sizeof(float), hipMemcpyHostToDevice);

	start = clock();
	/* run kernel */
 	for (int i=SIZE; i>1; i=i/2){
		Square_Diamond_4<<<ceil((float)N/256),256>>>(rng, (float*)dev_heightMap, i, dev_check1, dev_check2);
		hipDeviceSynchronize();
 	}
 	end = clock();

	/* memory copy from device to host*/
	hipMemcpy(heightMap, dev_heightMap, N * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(check1, dev_check1, N * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(check2, dev_check2, N * sizeof(float), hipMemcpyDeviceToHost);

	/* print the output */
	if(print){
		for (int i=0; i<N; i++){
		  printf("%d: x = %d, y = %d; hm = %f\n", i, i%(SIZE+1), i/(SIZE+1), heightMap[i]);
		}
	}
	// printf("%f\n", cpu_time_used);
	hipFree(dev_heightMap);
	hipFree(dev_check1);
	hipFree(dev_check2);

 	runTime = (double)(end - start)/CLOCKS_PER_SEC;
	printf("Run time for Version_4: %0.20f\n", runTime);
	return EXIT_SUCCESS;
}

int version_5 (void) {
	printf("5\n");
	return 0;
}


///////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////// VERSION 6.0 ////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////
/*	VERSION 6.0:
*			6. Less Threads Version: 1 * kernel + 1 * square device + 1 * diamond device (only active threads we need);
*			This kernel combine the four diamond kernel to one single kernel. However, each thread in diamond
*			kernel needs to calculate four vertex.
*/

/* square kernel to calculate the middle point */
__global__ void Square_6(hiprandState* rng, float* hm, int rect, float* check1, float* check2){
	/* set idx */
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int squareInRow = SIZE/rect;
  	if (idx < squareInRow * squareInRow){
		/* initialize vairable */
		int half = rect/2;
		int i, j, ni, nj, mi, mj;
	  
		/* calculate vertex */
		i = (idx%squareInRow*rect)%SIZE;
		j = (idx/squareInRow*rect)%SIZE;
		ni = i + rect;
		nj = j + rect;
		mi = i + half;
		mj = j + half;

		/* set check value */
		check1[idx] = mi;
		check2[idx] = mj;

		/* set random generator */
		float v1 = (0.0f - (float)ROUGHNESS)/2;
		float v2 = ((float)ROUGHNESS)/2;
		hiprandState localState = rng[idx];
	    float rand = v1 + (v2 - v1) * hiprand_uniform(&localState);
		rng[idx] = localState;

	    /* set height map */
		hm[mi + mj*(SIZE+1)] = (hm[i + j*(SIZE+1)] + hm[ni + j*(SIZE+1)] + hm[i + nj*(SIZE+1)] + hm[ni + nj*(SIZE+1)])/4 + rand;
		__syncthreads();
  	}
}

/* smart diamond kernel calculate the diamond vertex and each thread only calculate one vertex */
__global__ void Diamond_6(hiprandState* rng, float* hm, int rect, float* check1, float* check2){
	/* set idx */
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int squareInRow = SIZE/rect;
  	if (idx < 4 * squareInRow * squareInRow){
		/* initialize vairable */
		int half = rect/2;
		int i, j;
		int pmi, pmj;
		float hm_p;
		int num_p;
	  
		/* calculate vertex */
		i = (idx%squareInRow*rect)%SIZE;
		j = (idx/squareInRow*rect)%SIZE;

		/* Calculate the diamond vertex use idx */
		int tid = idx/(squareInRow*squareInRow)%4;
		pmi = i + (1 - tid%2)*half + tid/2*half;
		pmj = j + tid%2*half + tid/2*half;

		/* Set the value */
		hm_p = 0;
		num_p = 0;
		if (pmi - half >= 0){
			hm_p += hm[(pmi-half) + pmj*(SIZE+1)];
			num_p++;
		}
		if (pmi + half <= SIZE){
			hm_p += hm[(pmi+half) + pmj*(SIZE+1)];
			num_p++;
		}
		if (pmj - half >= 0){
			hm_p += hm[pmi + (pmj-half)*(SIZE+1)];
			num_p++;
		}
		if (pmj + half <= SIZE){
			hm_p += hm[pmi + (pmj+half)*(SIZE+1)];
			num_p++;
		}

		/* set check value */
		check1[idx] = pmi;
		check2[idx] = pmj;

		/* set random generator */
		float v1 = (0.0f - (float)ROUGHNESS)/2;
		float v2 = ((float)ROUGHNESS)/2;
		hiprandState localState = rng[idx];
	    float rand = v1 + (v2 - v1) * hiprand_uniform(&localState);

		/* get height for  */
		hm[pmi + pmj*(SIZE+1)] = hm_p/num_p + rand;
		rng[idx] = localState;
		__syncthreads();    
  	}
}

/* the host code for version 3: 1 square kernel + 1 smart diamond kernel */
int version_6 (void) {
	printf("Version 6: square kernel + smart diamond kernel (active less threads) \n");
	/* initialize variables */
	float check1[N];
	float check2[N];
	float heightMap[N];
	/* initialize device */
	float *dev_heightMap;
	float *dev_check1;
	float *dev_check2;
	/* initialize time*/
	clock_t start, end;
	double runTime;
	/* initial height map */
	for (int i=0; i<N; i++){
	  heightMap[i] = 0;
	}

	/* set height for corner */
	heightMap[0 + 0 * (SIZE+1)] = CORNER;
	printf("heightMap_corner0: %f\n", heightMap[0 + 0 * (SIZE+1)]);
	heightMap[SIZE + 0 * (SIZE+1)] = CORNER;
	printf("heightMap_corner1: %f\n", heightMap[SIZE + 0 * (SIZE+1)]);
	heightMap[0 + SIZE * (SIZE+1)] = CORNER;
	printf("heightMap_corner3: %f\n", heightMap[0 + SIZE * (SIZE+1)]);
	heightMap[SIZE + SIZE * (SIZE+1)] = CORNER;
	printf("heightMap_corner2: %f\n", heightMap[SIZE + SIZE * (SIZE+1)]);

	hiprandState* rng;
	/* allocate memory for device */
	hipMalloc(&rng, N * sizeof(hiprandState));
	hipMalloc((void**)&dev_heightMap, N * sizeof(float));
	hipMalloc((void**)&dev_check1, N * sizeof(float));
	hipMalloc((void**)&dev_check2, N * sizeof(float));

	/* memory copy from host to device */
	hipMemcpy(dev_heightMap, heightMap, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_check1, check1, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_check2, check2, N * sizeof(float), hipMemcpyHostToDevice);

	start = clock();
	/* run kernel */
 	for (int i=SIZE; i>1; i=i/2){
		Square_6<<<ceil((float)N/256),256>>>(rng, (float*)dev_heightMap, i, dev_check1, dev_check2);
		hipDeviceSynchronize();
		Diamond_6<<<ceil((float)N/256),256>>>(rng, (float*)dev_heightMap, i, dev_check1, dev_check2);
		hipDeviceSynchronize();
 	}
 	end = clock();

	/* memory copy from device to host*/
	hipMemcpy(heightMap, dev_heightMap, N * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(check1, dev_check1, N * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(check2, dev_check2, N * sizeof(float), hipMemcpyDeviceToHost);

	/* print the output */
	for (int i=0; i<N; i++){
	  printf("%d: x = %d, y = %d; hm = %f\n", i, i%(SIZE+1), i/(SIZE+1), heightMap[i]);
	}
	// printf("\n");	
	// for (int i=0; i<N; i++){
	//   printf("%d: pmi = %f, pmj = %f\n", i, check1[i], check2[i]);
	// }

	// printf("%f\n", cpu_time_used);
	hipFree(dev_heightMap);
	hipFree(dev_check1);
	hipFree(dev_check2);

 	runTime = (double)(end - start)/CLOCKS_PER_SEC;
	printf("Run time for Version_6: %0.20f\n", runTime);
	return EXIT_SUCCESS;
}


int version_7 (void) {
	printf("7\n");
	return 0;
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////// VERSION 8.0 ////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////
/*	VERSION 8.0:
*			8.0 Smarter Kernel Version: 1 * sqaure kernel + 1 * smart diamond kernel (1 thread => 1 vertex);
*			This version reconstruct the diamond kernel to use different threads for different vertx. Each 
*			thread in diamond kernel only need to calculate one vertex. (A simple revised 2D version of version 3)
*/
__global__ void Square_8(hiprandState* rng, float* hm, int rect, float* check1, float* check2){
	/* set idx */
	int idx_temp = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
  	if (idx_temp < SIZE+1 && idy < SIZE+1){
  		int idx = idy*(SIZE+1) + idx_temp;
		/* initialize vairable */
		int half = rect/2;
		int i, j, ni, nj, mi, mj;
		int squareInRow = SIZE/rect;
	  
		/* calculate vertex */
		i = (idx%squareInRow*rect)%SIZE;
		j = (idx/squareInRow*rect)%SIZE;
		ni = i + rect;
		nj = j + rect;
		mi = i + half;
		mj = j + half;

		/* set check value */
		check1[idx] = mi;
		check2[idx] = mj;

		/* set random generator */
		float v1 = (0.0f - (float)ROUGHNESS)/2;
		float v2 = ((float)ROUGHNESS)/2;
		hiprandState localState = rng[idx];
	    float rand = v1 + (v2 - v1) * hiprand_uniform(&localState);
		rng[idx] = localState;

	    /* set height map */
		hm[mi + mj*(SIZE+1)] = (hm[i + j*(SIZE+1)] + hm[ni + j*(SIZE+1)] + hm[i + nj*(SIZE+1)] + hm[ni + nj*(SIZE+1)])/4 +rand;
		__syncthreads();
  	}
}

__global__ void Diamond_8(hiprandState* rng, float* hm, int rect, float* check1, float* check2){
	/* set idx */
	int idx_temp = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
  	if (idx_temp < SIZE+1 && idy < SIZE+1){
  		int idx = idy*(SIZE+1) + idx_temp;
		/* initialize vairable */
		int half = rect/2;
		int i, j;
		int pmi, pmj;
		float hm_p;
		int num_p;
		int squareInRow = SIZE/rect;
	  
		/* calculate vertex */
		i = (idx%squareInRow*rect)%SIZE;
		j = (idx/squareInRow*rect)%SIZE;

		/* Calculate the diamond vertex use idx */
		int tid = idx/(squareInRow*squareInRow)%4;
		pmi = i + (1 - tid%2)*half + tid/2*half;
		pmj = j + tid%2*half + tid/2*half;

		/* Set the value */
		hm_p = 0;
		num_p = 0;
		if (pmi - half >= 0){
			hm_p += hm[(pmi-half) + pmj*(SIZE+1)];
			num_p++;
		}
		if (pmi + half <= SIZE){
			hm_p += hm[(pmi+half) + pmj*(SIZE+1)];
			num_p++;
		}
		if (pmj - half >= 0){
			hm_p += hm[pmi + (pmj-half)*(SIZE+1)];
			num_p++;
		}
		if (pmj + half <= SIZE){
			hm_p += hm[pmi + (pmj+half)*(SIZE+1)];
			num_p++;
		}

		/* set check value */
		check1[idx] = pmi;
		check2[idx] = pmj;

		/* set random generator */
		float v1 = (0.0f - (float)ROUGHNESS)/2;
		float v2 = ((float)ROUGHNESS)/2;
		hiprandState localState = rng[idx];
	    float rand = v1 + (v2 - v1) * hiprand_uniform(&localState);

		/* get height for  */
		hm[pmi + pmj*(SIZE+1)] = hm_p/num_p +rand;
		rng[idx] = localState;
		__syncthreads();    
  	}
}

/* the host code for version 8: 2D + 1 square kernel + 1 smart diamond kernel. */
int version_8 (bool print, int block_size) {
	printf("Version 8: square kernel + smart diamond kernel\n");
	/* initialize variables */
	float check1[N];
	float check2[N];
	float heightMap[N];
	/* initialize device */
	float *dev_heightMap;
	float *dev_check1;
	float *dev_check2;
	/* initialize time*/
	clock_t start, end;
	double runTime;
	/* initial height map */
	for (int i=0; i<N; i++){
	  heightMap[i] = 0;
	}

	/* set height for corner */
	heightMap[0 + 0 * (SIZE+1)] = 1;
	printf("heightMap_corner0: %f\n", heightMap[0 + 0 * (SIZE+1)]);
	heightMap[SIZE + 0 * (SIZE+1)] = 2;
	printf("heightMap_corner1: %f\n", heightMap[SIZE + 0 * (SIZE+1)]);
	heightMap[0 + SIZE * (SIZE+1)] = 3;
	printf("heightMap_corner3: %f\n", heightMap[0 + SIZE * (SIZE+1)]);
	heightMap[SIZE + SIZE * (SIZE+1)] = 4;
	printf("heightMap_corner2: %f\n", heightMap[SIZE + SIZE * (SIZE+1)]);

	hiprandState* rng;
	/* allocate memory for device */
	hipMalloc(&rng, N * sizeof(hiprandState));
	hipMalloc((void**)&dev_heightMap, N * sizeof(float));
	hipMalloc((void**)&dev_check1, N * sizeof(float));
	hipMalloc((void**)&dev_check2, N * sizeof(float));

	/* memory copy from host to device */
	hipMemcpy(dev_heightMap, heightMap, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_check1, check1, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_check2, check2, N * sizeof(float), hipMemcpyHostToDevice);

	start = clock();
	/* run kernel */
	dim3 DimGrid(ceil(((float)SIZE)/block_size),ceil(((float)SIZE)/block_size), 1); 
	dim3 DimBlock(block_size,block_size,1);
 	for (int i=SIZE; i>1; i=i/2){
		Square_8<<<DimGrid,DimBlock>>>(rng, (float*)dev_heightMap, i, dev_check1, dev_check2);
		hipDeviceSynchronize();
		Diamond_8<<<DimGrid,DimBlock>>>(rng, (float*)dev_heightMap, i, dev_check1, dev_check2);
		hipDeviceSynchronize();
 	}
 	end = clock();

	/* memory copy from device to host*/
	hipMemcpy(heightMap, dev_heightMap, N * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(check1, dev_check1, N * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(check2, dev_check2, N * sizeof(float), hipMemcpyDeviceToHost);

	/* print the output */
	if(print){
		for (int i=0; i<N; i++){
		  printf("%d: x = %d, y = %d; hm = %f\n", i, i%(SIZE+1), i/(SIZE+1), heightMap[i]);
		}
	}
	// printf("\n");	
//	for (int i=0; i<SIZE+1; i++){
//	   printf("%d: pmi = %f, pmj = %f\n", i, check1[i], check2[i]);
//	}

	// printf("%f\n", cpu_time_used);
	hipFree(dev_heightMap);
	hipFree(dev_check1);
	hipFree(dev_check2);

 	runTime = (double)(end - start)/CLOCKS_PER_SEC;
	printf("Run time for Version_8: %0.20f\n", runTime);
	return EXIT_SUCCESS;
}


///////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////// VERSION 9.0 ////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////
/*	VERSION 9.0:
*			9.0 Smarter Kernel Version: 1 * sqaure kernel + 1 * smart diamond kernel (1 thread => 1 vertex);
*			This version reconstruct the diamond kernel to use different threads for different vertx. Each 
*			thread in diamond kernel only need to calculate one vertex. (A simple revised 2D version of version 3)
*/
__global__ void Square_9(hiprandState* rng, float* hm, int rect, float* check1, float* check2){
	/* set idx */
	int idx_temp = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
  	if (idx_temp < SIZE+1 && idy < SIZE+1){
  		int idx = idy*(SIZE+1) + idx_temp;
		/* initialize vairable */
		int half = rect/2;
		int i, j, ni, nj, mi, mj;
		int squareInRow = SIZE/rect;
	  
		/* calculate vertex */
		i = (idx%squareInRow*rect)%SIZE;
		j = (idx/squareInRow*rect)%SIZE;
		ni = i + rect;
		nj = j + rect;
		mi = i + half;
		mj = j + half;

		/* set check value */
		check1[idx] = mi;
		check2[idx] = mj;

		/* set random generator */
		float v1 = (0.0f - (float)ROUGHNESS)/2;
		float v2 = ((float)ROUGHNESS)/2;
		hiprandState localState = rng[idx];
	    float rand = v1 + (v2 - v1) * hiprand_uniform(&localState);
		rng[idx] = localState;

	    /* set height map */
		hm[mi + mj*(SIZE+1)] = (hm[i + j*(SIZE+1)] + hm[ni + j*(SIZE+1)] + hm[i + nj*(SIZE+1)] + hm[ni + nj*(SIZE+1)])/4 +rand;
		__syncthreads();
  	}
}

__global__ void Diamond_9(hiprandState* rng, float* hm, int rect, float* check1, float* check2){
	/* set idx */
	int idx_temp = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
  	if (idx_temp < SIZE+1 && idy < SIZE+1){
  		int idx = idy*(SIZE+1) + idx_temp;
		/* initialize vairable */
		int half = rect/2;
		int i, j;
		int pmi, pmj;
		float hm_p;
		int num_p;
		int squareInRow = SIZE/rect;
	  
		/* calculate vertex */
		i = (idx%squareInRow*rect)%SIZE;
		j = (idx/squareInRow*rect)%SIZE;

		/* Calculate the diamond vertex use idx */
		int tid = idx/(squareInRow*squareInRow)%4;
		pmi = i + (1 - tid%2)*half + tid/2*half;
		pmj = j + tid%2*half + tid/2*half;

		/* Set the value */
		hm_p = 0;
		num_p = 0;
		if (pmi - half >= 0){
			hm_p += hm[(pmi-half) + pmj*(SIZE+1)];
			num_p++;
		}
		if (pmi + half <= SIZE){
			hm_p += hm[(pmi+half) + pmj*(SIZE+1)];
			num_p++;
		}
		if (pmj - half >= 0){
			hm_p += hm[pmi + (pmj-half)*(SIZE+1)];
			num_p++;
		}
		if (pmj + half <= SIZE){
			hm_p += hm[pmi + (pmj+half)*(SIZE+1)];
			num_p++;
		}

		/* set check value */
		check1[idx] = pmi;
		check2[idx] = pmj;

		/* set random generator */
		float v1 = (0.0f - (float)ROUGHNESS)/2;
		float v2 = ((float)ROUGHNESS)/2;
		hiprandState localState = rng[idx];
	    float rand = v1 + (v2 - v1) * hiprand_uniform(&localState);

		/* get height for  */
		hm[pmi + pmj*(SIZE+1)] = hm_p/num_p +rand;
		rng[idx] = localState;
		__syncthreads();    
  	}
}

/* the host code for version 8: 2D + 1 square kernel + 1 smart diamond kernel. */
int version_9 (bool print, int block_size) {
	printf("Version 8: square kernel + smart diamond kernel\n");
	/* initialize variables */
	float check1[N];
	float check2[N];
	float heightMap[N];
	/* initialize device */
	float *dev_heightMap;
	float *dev_check1;
	float *dev_check2;
	/* initialize time*/
	clock_t start, end;
	double runTime;
	/* initial height map */
	for (int i=0; i<N; i++){
	  heightMap[i] = 0;
	}

	/* set height for corner */
	heightMap[0 + 0 * (SIZE+1)] = 1;
	printf("heightMap_corner0: %f\n", heightMap[0 + 0 * (SIZE+1)]);
	heightMap[SIZE + 0 * (SIZE+1)] = 2;
	printf("heightMap_corner1: %f\n", heightMap[SIZE + 0 * (SIZE+1)]);
	heightMap[0 + SIZE * (SIZE+1)] = 3;
	printf("heightMap_corner3: %f\n", heightMap[0 + SIZE * (SIZE+1)]);
	heightMap[SIZE + SIZE * (SIZE+1)] = 4;
	printf("heightMap_corner2: %f\n", heightMap[SIZE + SIZE * (SIZE+1)]);

	hiprandState* rng;
	/* allocate memory for device */
	hipMalloc(&rng, N * sizeof(hiprandState));
	hipMalloc((void**)&dev_heightMap, N * sizeof(float));
	hipMalloc((void**)&dev_check1, N * sizeof(float));
	hipMalloc((void**)&dev_check2, N * sizeof(float));

	/* memory copy from host to device */
	hipMemcpy(dev_heightMap, heightMap, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_check1, check1, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_check2, check2, N * sizeof(float), hipMemcpyHostToDevice);

	start = clock();
	/* run kernel */
	dim3 DimGrid(ceil(((float)SIZE)/block_size),ceil(((float)SIZE)/block_size), 1); 
	dim3 DimBlock(block_size,block_size,1);
 	for (int i=SIZE; i>1; i=i/2){
		Square_9<<<DimGrid,DimBlock>>>(rng, (float*)dev_heightMap, i, dev_check1, dev_check2);
		hipDeviceSynchronize();
		Diamond_9<<<DimGrid,DimBlock>>>(rng, (float*)dev_heightMap, i, dev_check1, dev_check2);
		hipDeviceSynchronize();
 	}
 	end = clock();

	/* memory copy from device to host*/
	hipMemcpy(heightMap, dev_heightMap, N * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(check1, dev_check1, N * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(check2, dev_check2, N * sizeof(float), hipMemcpyDeviceToHost);

	/* print the output */
	if(print){
		for (int i=0; i<N; i++){
		  printf("%d: x = %d, y = %d; hm = %f\n", i, i%(SIZE+1), i/(SIZE+1), heightMap[i]);
		}
	}
	// printf("\n");	
//	for (int i=0; i<SIZE+1; i++){
//	   printf("%d: pmi = %f, pmj = %f\n", i, check1[i], check2[i]);
//	}

	// printf("%f\n", cpu_time_used);
	hipFree(dev_heightMap);
	hipFree(dev_check1);
	hipFree(dev_check2);

 	runTime = (double)(end - start)/CLOCKS_PER_SEC;
	printf("Run time for Version_8: %0.20f\n", runTime);
	return EXIT_SUCCESS;
}

#include "hip/hip_runtime.h"
/************************************************************
*	ECE408 Parallel Programming - Final Project				*
*															*
*	Topic: Terrain Generation								*
*	Members: Lai,Haoming; Ma,Yunhan; Wang,Bangqi			*
*															*
************************************************************/

/*
* Terrain Generation:
* Algorithmn: Diamond Square Algorithmn.
* Version:
*			0. Serial version: 1 * square loop + 4 * diamond loop;
*			1. Parallel version: 1 * sdsfsdfsdf + 4 * diamond kernel;
*			2. Less Kernel Version: 1 * square kernal + 1 * simple diamond kernel (1 thread => 4 vertex);
*			3. Smarter Kernel Version: 1 * sqaure kernel + 1 * smart diamond kernel (1 thread => 1 vertex);
*			4. One Kernel Version: 1 * square_diamond kernel combined; (based on version 2)
*			5. Kernel Device Version: 1 * kernel + 1 * square device + 1 * diamond device;
*			6. Less Threads Version: 1 * kernel + 1 * square device + 1 * diamond device (only active threads we need);
*			7. Less Block Version: 1 * kernel + 1 * square device + 1 * diamond device (only call the size of kernel we need);
*			8. Shared Memory Version: 1 * kernel + 1 * square device + 1 * diamond device (use share memory);
*			9. 2D Simple Kernel Version: 1 * sqaure kernel + 1 * smart diamond kernel (1 thread => 1 vertex)(based on version 3);
*			10. 2D Smarter Kernel Version: 1 * sqaure kernel + 1 * smart diamond kernel (1 thread => 1 vertex)(based on version 6);
*			11. 2D Smarter, Less Threads, less block Version: 1 * sqaure kernel + 1 * smart diamond kernel (1 thread => 1 vertex)(based on version 10);
*		
*/


#include "hip/hip_runtime.h"
#include ""
#include <hip/device_functions.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <math.h>

/* Set the parameter */
/* Choose the version to use */
//#define VERSION 0
/* Set the length of each edge. please put power of 2 */
#define SIZE 4096
/* Set number of array */
#define N (SIZE+1)*(SIZE+1)
/* Set the roughness for terrain */
#define ROUGHNESS 4
/* Set the height for each corner */
#define CORNER 0

/* main function for different version */
int version_0(void);
int version_1(int block_size);
int version_2(int block_size);
int version_3(bool print, int block_size);
int version_4(bool print, int block_size);
int version_5(bool print, int block_size);
int version_6(bool print, int block_size);
int version_7(bool print, int block_size);
int version_8(void);
int version_9(bool print, int block_size);
int version_10(bool print, int block_size);
int version_11(bool print, int block_size);
int version_12(bool print, int block_size);
int version_100(bool print);

/* main function */
int main(void){
	int VERSION;
	int p;
	int block_size;
	int cont =1;
	
	while (cont == 1){
		bool print = false;
		printf("what version do you want: ");
		scanf("%d", &VERSION);
		printf("print? (0/1): ");
		scanf("%d", &p);
		printf("please define block_size(max = 32): ");
		scanf("%d", &block_size);
		if (p)
			print = true;

		switch (VERSION){
			/* test version 0 */
		case 0:
			version_0();
			break;
		case 1:
			/* test version 1 */
			version_1(block_size);
			break;
		case 2:
			/* test version 2 */
			version_2(block_size);
			break;
		case 3:
			/* test version 3 */
			version_3(print, block_size);
			break;
		case 4:
			/* test version 4 */
			version_4(print, block_size);
			break;
		case 5:
			/* test version 5 */
			version_5(print, block_size);
			break;
		case 6:
			/* test version 6 */
			version_6(print, block_size);
			break;
		case 7:
			/* test version 7 */
			version_7(print, block_size);
			break;
		case 8:
			/* test version 8 */
			version_8();
			break;
		case 9:
			/* test version 9 */
			version_9(print, block_size);
			break;
		case 10:
			/* test version 10 */
			version_10(print, block_size);
			break;
		case 11:
			/* test version 10 */
			version_11(print, block_size);
			break;
		case 12:
			/* test version 10 */
			version_12(print, block_size);
			break;
		case 100:
			/* test version 10 */
			version_100(print);
			break;
		default:
			/* test version 0 */
			version_0();
			return 0;
		}
		printf("done. Continue(1=continue, other#=exit)?");
		scanf("%d", &cont);
	}
}


///////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////// VERSION 0.0 ////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////
/*	VERSION 0.0:
*			0.0 Serial version: 1 * square loop + 4 * loop;
*
*/

/* host code for serial version */
int version_0(void) {
	clock_t start, end;
	double runTime;
	float **heightMap = new float*[SIZE + 1];
	for (int i = 0; i<SIZE + 1; i++)
		heightMap[i] = new float[SIZE + 1];

	for (int i = 0; i<SIZE + 1; i++){
		for (int j = 0; j<SIZE + 1; j++){
			heightMap[i][j] = 0.0;
		}
	}

	//initial the first four points
	heightMap[0][0] = 0;
	heightMap[SIZE][0] = 0;
	heightMap[0][SIZE] = 0;
	heightMap[SIZE][SIZE] = 0;
	srand(time(NULL));

	start = clock();
	int stride = SIZE;
	while (stride >= 2){
		for (int i = 0; i<(SIZE / stride); i++){
			for (int j = 0; j<(SIZE / stride); j++){
				int leftbottom_x = i* stride;
				int leftbottom_y = j* stride;
				float average = heightMap[leftbottom_x][leftbottom_y] + heightMap[leftbottom_x + stride][leftbottom_y] + heightMap[leftbottom_x][leftbottom_y + stride] + heightMap[leftbottom_x + stride][leftbottom_y + stride];
				average = average / 4;
				float guess = ((float)((rand() % 200) - 100) / 50);
		//		printf("%1.5f\n", guess);
				heightMap[leftbottom_x + stride / 2][leftbottom_y + stride / 2] = average + ((float)stride/128)*guess;

				heightMap[leftbottom_x + stride / 2][leftbottom_y] = (average + heightMap[leftbottom_x][leftbottom_y] + heightMap[leftbottom_x + stride][leftbottom_y]) / 3 + ((float)stride / 128)*guess;
				heightMap[leftbottom_x][leftbottom_y + stride / 2] = (average + heightMap[leftbottom_x][leftbottom_y] + heightMap[leftbottom_x][leftbottom_y + stride]) / 3 + ((float)stride / 128)*guess;
				heightMap[leftbottom_x + stride][leftbottom_y + stride / 2] = (average + heightMap[leftbottom_x + stride][leftbottom_y] + heightMap[leftbottom_x + stride][leftbottom_y + stride]) / 3 + ((float)stride / 128)*guess;
				heightMap[leftbottom_x + stride / 2][leftbottom_y + stride] = (average + heightMap[leftbottom_x][leftbottom_y + stride] + heightMap[leftbottom_x + stride][leftbottom_y + stride]) / 3 + ((float)stride / 128)*guess;

			}
		}
		printf("%d \n", stride);
		stride = stride / 2;
	}


	end = clock();
	runTime = (double)(end - start) / CLOCKS_PER_SEC;

	// 	for (int i=0; i<=SIZE; i++){
	// 		for(int j=0; j<=SIZE; j++){
	//	 		printf("%d: x = %d, y = %d; hm = %f\n", i*j, i, j, heightMap[i][j]);
	//		}
	//	}

	FILE *fp_out;
	float index_x;
	float index_y;
	fp_out = fopen("vertex.txt", "w");
	for (int j = 0; j<SIZE + 1; j++){
		for (int i = 0; i<SIZE + 1; i++){
			index_x = (float)i / (SIZE / 2) - 1;
			index_y = (float)j / (SIZE / 2) - 1;
			fprintf(fp_out, "%f %f %f\n", index_x, index_y, heightMap[i][j]);
		}
	}
	fclose(fp_out);
	printf("Run time for Version_0: %f\n", runTime);
	printf("Version 0\n");
	for (int i = SIZE; i >= 0; i--)
		delete[] heightMap[i];
	delete[] heightMap;
	return 0;
}



///////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////// VERSION 1.0 ////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////
/*	VERSION 1.0:
*			1.0 Parallel version: 1 * square kernel + 4 * diamond kernel;
*			This parallel function parallelize the serial code directly. it change the one square loop to
*			one square kernel and change four diamond loop to four different diamond kernel.	1
*/

/* setup random number*/
__global__ void setseed(hiprandState * state, unsigned long seed)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	hiprand_init(seed, id, 0, &state[id]);
}

__global__ void generate(float* random, hiprandState* globalState, int n)
{
	// generate random numbers
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx<n)
	{
		hiprandState localState = globalState[idx];
		float RANDOM = hiprand_uniform(&localState);
		globalState[idx] = localState;
		random[idx] = RANDOM;
	}
}


/* square kernel to calculate the middle point */
__global__ void Square_1(hiprandState* rng, float* hm, int rect){
	/* set idx */
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < N){
		/* initialize vairable */
		int half = rect / 2;
		int i, j, ni, nj, mi, mj;
		int squareInRow = SIZE / rect;

		/* calculate vertex */
		i = (idx%squareInRow*rect) % SIZE;
		j = (idx / squareInRow*rect) % SIZE;
		ni = i + rect;
		nj = j + rect;
		mi = i + half;
		mj = j + half;

		/* set random generator */
		float v1 = (0.0f - (float)ROUGHNESS) / 2;
		float v2 = ((float)ROUGHNESS) / 2;
		hiprandState localState = rng[idx];
		float rand = v1 + (v2 - v1) * hiprand_uniform(&localState);
		rng[idx] = localState;

		/* set height map */
		hm[mi + mj*(SIZE + 1)] = (hm[i + j*(SIZE + 1)] + hm[ni + j*(SIZE + 1)] + hm[i + nj*(SIZE + 1)] + hm[ni + nj*(SIZE + 1)]) / 4 + rand;
		__syncthreads();
	}
}

/* diamond kernel 1_1 to calcualte middle bottom point */
__global__ void Diamond_1_1(hiprandState* rng, float* hm, int rect){
	/* set idx */
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < N){
		/* initialize vairable */
		int half = rect / 2;
		int i, mi, j;
		int pmi_b, pmj_b;
		float hm_b;
		int num_b;
		int squareInRow = SIZE / rect;

		/* calculate vertex */
		i = (idx%squareInRow*rect) % SIZE;
		j = (idx / squareInRow*rect) % SIZE;
		mi = i + half;

		/* find 4 diamond vertex */
		/* bottom vertex */
		pmi_b = mi;
		pmj_b = j;

		/* set the value */
		/* bottom height */
		hm_b = 0;
		num_b = 3;
		if (pmj_b - half >= 0){
			hm_b += hm[pmi_b + (pmj_b - half)*(SIZE + 1)];
			num_b = 4;
		}
		hm_b += hm[pmi_b + (pmj_b + half)*(SIZE + 1)];
		hm_b += hm[(pmi_b - half) + pmj_b*(SIZE + 1)];
		hm_b += hm[(pmi_b + half) + pmj_b*(SIZE + 1)];

		/* set random generator */
		float v1 = (0.0f - (float)ROUGHNESS) / 2;
		float v2 = ((float)ROUGHNESS) / 2;
		hiprandState localState = rng[idx];
		float rand1 = v1 + (v2 - v1) * hiprand_uniform(&localState);

		/* set height map */
		hm[pmi_b + pmj_b*(SIZE + 1)] = hm_b / num_b + rand1;
		// hm[mi + mj*(SIZE+1)] = (hm[i + j*(SIZE+1)] + hm[ni + j*(SIZE+1)] + hm[i + nj*(SIZE+1)] + hm[ni + nj*(SIZE+1)])/4 + rand;
		rng[idx] = localState;
		__syncthreads();
	}
}

/* diamond kernel 1_2 to calcualte left point */
__global__ void Diamond_1_2(hiprandState* rng, float* hm, int rect){
	/* set idx */
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < N){
		/* initialize vairable */
		int half = rect / 2;
		int i, j, mj;
		int pmi_l, pmj_l;
		float hm_l;
		int num_l;
		int squareInRow = SIZE / rect;

		/* calculate vertex */
		i = (idx%squareInRow*rect) % SIZE;
		j = (idx / squareInRow*rect) % SIZE;
		mj = j + half;

		/* find 4 diamond vertex */
		/* left vertex */
		pmi_l = i;
		pmj_l = mj;

		/* set the value */
		/* left height */
		hm_l = 0;
		num_l = 3;
		if (pmi_l - half >= 0){
			hm_l += hm[(pmi_l - half) + pmj_l*(SIZE + 1)];
			num_l = 4;
		}
		hm_l += hm[(pmi_l + half) + pmj_l*(SIZE + 1)];
		hm_l += hm[pmi_l + (pmj_l - half)*(SIZE + 1)];
		hm_l += hm[pmi_l + (pmj_l + half)*(SIZE + 1)];

		/* set random generator */
		float v1 = (0.0f - (float)ROUGHNESS) / 2;
		float v2 = ((float)ROUGHNESS) / 2;
		hiprandState localState = rng[idx];
		float rand2 = v1 + (v2 - v1) * hiprand_uniform(&localState);

		/* set height map */
		hm[pmi_l + pmj_l*(SIZE + 1)] = hm_l / num_l + rand2;

		// hm[mi + mj*(SIZE+1)] = (hm[i + j*(SIZE+1)] + hm[ni + j*(SIZE+1)] + hm[i + nj*(SIZE+1)] + hm[ni + nj*(SIZE+1)])/4 + rand;
		rng[idx] = localState;
		__syncthreads();
	}
}

/* diamond kernel 1_3 to calcualte right point */
__global__ void Diamond_1_3(hiprandState* rng, float* hm, int rect){
	/* set idx */
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < N){
		/* initialize vairable */
		int half = rect / 2;
		int i, j, ni, mj;
		int pmi_r, pmj_r;
		float hm_r;
		int num_r;
		int squareInRow = SIZE / rect;

		/* calculate vertex */
		i = (idx%squareInRow*rect) % SIZE;
		j = (idx / squareInRow*rect) % SIZE;
		ni = i + rect;
		mj = j + half;

		/* find 4 diamond vertex */
		/* right vertex */
		pmi_r = ni;
		pmj_r = mj;

		/* set the value */
		/* right height */
		hm_r = 0;
		num_r = 3;
		if (pmi_r + half <= SIZE){
			hm_r += hm[(pmi_r + half) + pmj_r*(SIZE + 1)];
			num_r = 4;
		}
		hm_r += hm[(pmi_r - half) + pmj_r*(SIZE + 1)];
		hm_r += hm[pmi_r + (pmj_r - half)*(SIZE + 1)];
		hm_r += hm[pmi_r + (pmj_r + half)*(SIZE + 1)];


		/* get height for  */

		/* set random generator */
		float v1 = (0.0f - (float)ROUGHNESS) / 2;
		float v2 = ((float)ROUGHNESS) / 2;
		hiprandState localState = rng[idx];
		float rand3 = v1 + (v2 - v1) * hiprand_uniform(&localState);

		/* set height map */
		hm[pmi_r + pmj_r*(SIZE + 1)] = hm_r / num_r + rand3;
		// hm[mi + mj*(SIZE+1)] = (hm[i + j*(SIZE+1)] + hm[ni + j*(SIZE+1)] + hm[i + nj*(SIZE+1)] + hm[ni + nj*(SIZE+1)])/4 + rand;
		rng[idx] = localState;
		__syncthreads();
	}
}

/* diamond kernel 1_4 to calcualte middle top point */
__global__ void Diamond_1_4(hiprandState* rng, float* hm, int rect){
	/* set idx */
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < N){
		/* initialize vairable */
		int half = rect / 2;
		int i, j, mi, nj;
		int pmi_t, pmj_t;
		float hm_t;
		int num_t;
		int squareInRow = SIZE / rect;

		/* calculate vertex */
		i = (idx%squareInRow*rect) % SIZE;
		j = (idx / squareInRow*rect) % SIZE;
		nj = j + rect;
		mi = i + half;

		/* find 4 diamond vertex */
		/* top vertex */
		pmi_t = mi;
		pmj_t = nj;

		/* set the value */
		/* top height */
		hm_t = 0;
		num_t = 3;
		if (pmj_t + half <= SIZE){
			hm_t += hm[pmi_t + (pmj_t + half)*(SIZE + 1)];
			num_t = 4;
		}
		hm_t += hm[pmi_t + (pmj_t - half)*(SIZE + 1)];
		hm_t += hm[(pmi_t - half) + pmj_t*(SIZE + 1)];
		hm_t += hm[(pmi_t + half) + pmj_t*(SIZE + 1)];

		/* get height for  */

		/* set random generator */
		float v1 = (0.0f - (float)ROUGHNESS) / 2;
		float v2 = ((float)ROUGHNESS) / 2;
		hiprandState localState = rng[idx];
		float rand4 = v1 + (v2 - v1) * hiprand_uniform(&localState);

		/* set height map */
		hm[pmi_t + pmj_t*(SIZE + 1)] = hm_t / num_t + rand4;
		// hm[mi + mj*(SIZE+1)] = (hm[i + j*(SIZE+1)] + hm[ni + j*(SIZE+1)] + hm[i + nj*(SIZE+1)] + hm[ni + nj*(SIZE+1)])/4 + rand;
		rng[idx] = localState;
		__syncthreads();
	}
}

/* host code for version 1.0 */
int version_1(int block_size) {
	printf("Version 1: square kernel + 4 diamond kernel\n");
	/* initialize variables */
	float *heightMap = new float[N];
	/* initialize device */
	float *dev_heightMap;
	/* initialize time*/
	clock_t start, end;
	double runTime;
	/* initial height map */
	for (int i = 0; i<N; i++){
		heightMap[i] = 0;
	}

	/* set height for corner */
	heightMap[0 + 0 * (SIZE + 1)] = CORNER;
	printf("heightMap_corner0: %f\n", heightMap[0 + 0 * (SIZE + 1)]);
	heightMap[SIZE + 0 * (SIZE + 1)] = CORNER;
	printf("heightMap_corner1: %f\n", heightMap[SIZE + 0 * (SIZE + 1)]);
	heightMap[0 + SIZE * (SIZE + 1)] = CORNER;
	printf("heightMap_corner3: %f\n", heightMap[0 + SIZE * (SIZE + 1)]);
	heightMap[SIZE + SIZE * (SIZE + 1)] = CORNER;
	printf("heightMap_corner2: %f\n", heightMap[SIZE + SIZE * (SIZE + 1)]);

	hiprandState* rng;
	/* allocate memory for device */
	hipMalloc(&rng, N * sizeof(hiprandState));
	hipMalloc((void**)&dev_heightMap, N * sizeof(float));

	/* memory copy from host to device */
	hipMemcpy(dev_heightMap, heightMap, N * sizeof(float), hipMemcpyHostToDevice);

	start = clock();
	/* run kernel */
	for (int i = SIZE; i>1; i = i / 2){
		Square_1 << <ceil((float)N / (block_size*block_size)), (block_size*block_size) >> >(rng, (float*)dev_heightMap, i);
		hipDeviceSynchronize();
		Diamond_1_1 << <ceil((float)N / (block_size*block_size)), (block_size*block_size) >> >(rng, (float*)dev_heightMap, i);
		hipDeviceSynchronize();
		Diamond_1_2 << <ceil((float)N / (block_size*block_size)), (block_size*block_size) >> >(rng, (float*)dev_heightMap, i);
		hipDeviceSynchronize();
		Diamond_1_3 << <ceil((float)N / (block_size*block_size)), (block_size*block_size) >> >(rng, (float*)dev_heightMap, i);
		hipDeviceSynchronize();
		Diamond_1_4 << <ceil((float)N / (block_size*block_size)), (block_size*block_size) >> >(rng, (float*)dev_heightMap, i);
		hipDeviceSynchronize();
	}
	end = clock();

	/* memory copy from device to host*/
	hipMemcpy(heightMap, dev_heightMap, N * sizeof(float), hipMemcpyDeviceToHost);

	/* print the output */
//	for (int i = 0; i<N; i++){
//		printf("%d: x = %d, y = %d; hm = %f\n", i, i % (SIZE + 1), i / (SIZE + 1), heightMap[i]);
//	}

	// printf("%f\n", cpu_time_used);
	hipFree(dev_heightMap);

	runTime = (double)(end - start) / CLOCKS_PER_SEC;
	printf("Run time for Version_1: %f\n", runTime);

/*	FILE *fp_out;
	fp_out = fopen("vertex.txt", "w");
	for (int i = 0; i<N; i++)
		fprintf(fp_out, "%d %d %f\n", i % (SIZE + 1), i / (SIZE + 1), heightMap[i]);
	fclose(fp_out);
*/	
	delete[] heightMap;
	return EXIT_SUCCESS;
}




///////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////// VERSION 2.0 ////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////
/*	VERSION 2.0:
*			2.0 Less Kernel Version: 1 * square kernal + 1 * simple diamond kernel (1 thread => 4 vertex);
*			This kernel combine the four diamond kernel to one single kernel. However, each thread in diamond
*			kernel needs to calculate four vertex.
*/

/* combined diamond kernel to calculate the four point in each thread */
__global__ void Diamond_2(hiprandState* rng, float* hm, int rect){
	/* set idx */
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < N){
		/* initialize vairable */
		int half = rect / 2;
		int i, j, ni, nj, mi, mj;
		int pmi_b, pmj_b, pmi_l, pmj_l, pmi_r, pmj_r, pmi_t, pmj_t;
		float hm_b, hm_l, hm_r, hm_t;
		int num_b, num_l, num_r, num_t;
		int squareInRow = SIZE / rect;

		/* calculate vertex */
		i = (idx%squareInRow*rect) % SIZE;
		j = (idx / squareInRow*rect) % SIZE;
		ni = i + rect;
		nj = j + rect;
		mi = i + half;
		mj = j + half;

		/* find 4 diamond vertex */
		/* bottom vertex */
		pmi_b = mi;
		pmj_b = j;
		/* left vertex */
		pmi_l = i;
		pmj_l = mj;
		/* right vertex */
		pmi_r = ni;
		pmj_r = mj;
		/* top vertex */
		pmi_t = mi;
		pmj_t = nj;

		/* set the value */
		/* bottom height */
		hm_b = 0;
		num_b = 3;
		if (pmj_b - half >= 0){
			hm_b += hm[pmi_b + (pmj_b - half)*(SIZE + 1)];
			num_b = 4;
		}
		hm_b += hm[pmi_b + (pmj_b + half)*(SIZE + 1)];
		hm_b += hm[(pmi_b - half) + pmj_b*(SIZE + 1)];
		hm_b += hm[(pmi_b + half) + pmj_b*(SIZE + 1)];

		/* left height */
		hm_l = 0;
		num_l = 3;
		if (pmi_l - half >= 0){
			hm_l += hm[(pmi_l - half) + pmj_l*(SIZE + 1)];
			num_l = 4;
		}
		hm_l += hm[(pmi_l + half) + pmj_l*(SIZE + 1)];
		hm_l += hm[pmi_l + (pmj_l - half)*(SIZE + 1)];
		hm_l += hm[pmi_l + (pmj_l + half)*(SIZE + 1)];

		/* right height */
		hm_r = 0;
		num_r = 3;
		if (pmi_r + half <= SIZE){
			hm_r += hm[(pmi_r + half) + pmj_r*(SIZE + 1)];
			num_r = 4;
		}
		hm_r += hm[(pmi_r - half) + pmj_r*(SIZE + 1)];
		hm_r += hm[pmi_r + (pmj_r - half)*(SIZE + 1)];
		hm_r += hm[pmi_r + (pmj_r + half)*(SIZE + 1)];

		/* top height */
		hm_t = 0;
		num_t = 3;
		if (pmj_t + half <= SIZE){
			hm_t += hm[pmi_t + (pmj_t + half)*(SIZE + 1)];
			num_t = 4;
		}
		hm_t += hm[pmi_t + (pmj_t - half)*(SIZE + 1)];
		hm_t += hm[(pmi_t - half) + pmj_t*(SIZE + 1)];
		hm_t += hm[(pmi_t + half) + pmj_t*(SIZE + 1)];


		/* get height for  */

		/* set random generator */
		float v1 = (0.0f - (float)ROUGHNESS) / 2;
		float v2 = ((float)ROUGHNESS) / 2;
		hiprandState localState = rng[idx];
		float rand1 = v1 + (v2 - v1) * hiprand_uniform(&localState);
		float rand2 = v1 + (v2 - v1) * hiprand_uniform(&localState);
		float rand3 = v1 + (v2 - v1) * hiprand_uniform(&localState);
		float rand4 = v1 + (v2 - v1) * hiprand_uniform(&localState);

		/* set height map */
		hm[pmi_b + pmj_b*(SIZE + 1)] = hm_b / num_b + rand1;
		hm[pmi_l + pmj_l*(SIZE + 1)] = hm_l / num_l + rand2;
		hm[pmi_r + pmj_r*(SIZE + 1)] = hm_r / num_r + rand3;
		hm[pmi_t + pmj_t*(SIZE + 1)] = hm_t / num_t + rand4;
		// hm[mi + mj*(SIZE+1)] = (hm[i + j*(SIZE+1)] + hm[ni + j*(SIZE+1)] + hm[i + nj*(SIZE+1)] + hm[ni + nj*(SIZE+1)])/4 + rand;
		rng[idx] = localState;
		__syncthreads();
	}
}


/* the host code for version 2: 1 square kernel + 1 stupid diamond kernel */
int version_2(int block_size) {
	printf("Version 2: square kernel + stupid diamond kernel\n");
	/* initialize variables */
	float *heightMap= new float[N];
	/* initialize device */
	float *dev_heightMap;
	/* initialize time*/
	clock_t start, end;
	double runTime;
	/* initial height map */
	for (int i = 0; i<N; i++){
		heightMap[i] = 0;
	}

	/* set height for corner */
	heightMap[0 + 0 * (SIZE + 1)] = CORNER;
	printf("heightMap_corner0: %f\n", heightMap[0 + 0 * (SIZE + 1)]);
	heightMap[SIZE + 0 * (SIZE + 1)] = CORNER;
	printf("heightMap_corner1: %f\n", heightMap[SIZE + 0 * (SIZE + 1)]);
	heightMap[0 + SIZE * (SIZE + 1)] = CORNER;
	printf("heightMap_corner3: %f\n", heightMap[0 + SIZE * (SIZE + 1)]);
	heightMap[SIZE + SIZE * (SIZE + 1)] = CORNER;
	printf("heightMap_corner2: %f\n", heightMap[SIZE + SIZE * (SIZE + 1)]);

	hiprandState* rng;
	/* allocate memory for device */
	hipMalloc(&rng, N * sizeof(hiprandState));
	hipMalloc((void**)&dev_heightMap, N * sizeof(float));

	/* memory copy from host to device */
	hipMemcpy(dev_heightMap, heightMap, N * sizeof(float), hipMemcpyHostToDevice);

	start = clock();
	/* run kernel */
	for (int i = SIZE; i>1; i = i / 2){
		Square_1 << <ceil((float)N / (block_size*block_size)), (block_size*block_size) >> >(rng, (float*)dev_heightMap, i);
		hipDeviceSynchronize();
		Diamond_2 << <ceil((float)N / (block_size*block_size)), (block_size*block_size) >> >(rng, (float*)dev_heightMap, i);
		hipDeviceSynchronize();
	}
	end = clock();

	/* memory copy from device to host*/
	hipMemcpy(heightMap, dev_heightMap, N * sizeof(float), hipMemcpyDeviceToHost);

	/* print the output */
//	for (int i = 0; i<N; i++){
//		printf("%d: x = %d, y = %d; hm = %f\n", i, i % (SIZE + 1), i / (SIZE + 1), heightMap[i]);
//	}

	// printf("%f\n", cpu_time_used);
	hipFree(dev_heightMap);

	runTime = (double)(end - start) / CLOCKS_PER_SEC;
	printf("Run time for Version_2: %0.20f\n", runTime);
	delete[] heightMap;
	return EXIT_SUCCESS;
}




///////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////// VERSION 3.0 ////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////
/*	VERSION 3.0:
*			3.0 Smarter Kernel Version: 1 * sqaure kernel + 1 * smart diamond kernel (1 thread => 1 vertex);
*			This version reconstruct the diamond kernel to use different threads for different vertx. Each
*			thread in diamond kernel only need to calculate one vertex.
*/

/* smart diamond kernel calculate the diamond vertex and each thread only calculate one vertex */
__global__ void Diamond_3(hiprandState* rng, float* hm, int rect){
	/* set idx */
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < N){
		/* initialize vairable */
		int half = rect / 2;
		int i, j;
		int pmi, pmj;
		float hm_p;
		int num_p;
		int squareInRow = SIZE / rect;

		/* calculate vertex */
		i = (idx%squareInRow*rect) % SIZE;
		j = (idx / squareInRow*rect) % SIZE;

		/* Calculate the diamond vertex use idx */
		int tid = idx / (squareInRow*squareInRow) % 4;
		pmi = i + (1 - tid % 2)*half + tid / 2 * half;
		pmj = j + tid % 2 * half + tid / 2 * half;

		/* Set the value */
		hm_p = 0;
		num_p = 0;
		if (pmi - half >= 0){
			hm_p += hm[(pmi - half) + pmj*(SIZE + 1)];
			num_p++;
		}
		if (pmi + half <= SIZE){
			hm_p += hm[(pmi + half) + pmj*(SIZE + 1)];
			num_p++;
		}
		if (pmj - half >= 0){
			hm_p += hm[pmi + (pmj - half)*(SIZE + 1)];
			num_p++;
		}
		if (pmj + half <= SIZE){
			hm_p += hm[pmi + (pmj + half)*(SIZE + 1)];
			num_p++;
		}


		/* set random generator */
		float v1 = (0.0f - (float)ROUGHNESS) / 2;
		float v2 = ((float)ROUGHNESS) / 2;
		hiprandState localState = rng[idx];
		float rand = v1 + (v2 - v1) * hiprand_uniform(&localState);

		/* get height for  */
		hm[pmi + pmj*(SIZE + 1)] = hm_p / num_p + rand;
		rng[idx] = localState;
		__syncthreads();
	}
}

/* the host code for version 3: 1 square kernel + 1 smart diamond kernel */
int version_3(bool print, int block_size) {
	printf("Version 3: square kernel + smart diamond kernel\n");
	/* initialize variables */
	float *heightMap = new float[N];
	/* initialize device */
	float *dev_heightMap;

	/* initialize time*/
	clock_t start, end;
	double runTime;
	int size = block_size * block_size;
	/* initial height map */
	for (int i = 0; i<N; i++){
		heightMap[i] = 0;
	}

	/* set height for corner */
	heightMap[0 + 0 * (SIZE + 1)] = CORNER;
	printf("heightMap_corner0: %f\n", heightMap[0 + 0 * (SIZE + 1)]);
	heightMap[SIZE + 0 * (SIZE + 1)] = CORNER;
	printf("heightMap_corner1: %f\n", heightMap[SIZE + 0 * (SIZE + 1)]);
	heightMap[0 + SIZE * (SIZE + 1)] = CORNER;
	printf("heightMap_corner3: %f\n", heightMap[0 + SIZE * (SIZE + 1)]);
	heightMap[SIZE + SIZE * (SIZE + 1)] = CORNER;
	printf("heightMap_corner2: %f\n", heightMap[SIZE + SIZE * (SIZE + 1)]);

	hiprandState* rng;
	/* allocate memory for device */
	hipMalloc(&rng, N * sizeof(hiprandState));
	hipMalloc((void**)&dev_heightMap, N * sizeof(float));

	/* memory copy from host to device */
	hipMemcpy(dev_heightMap, heightMap, N * sizeof(float), hipMemcpyHostToDevice);

	start = clock();
	/* run kernel */
	for (int i = SIZE; i>1; i = i / 2){
		Square_1 << <ceil((float)N / (block_size*block_size)), (block_size*block_size) >> >(rng, (float*)dev_heightMap, i);
		hipDeviceSynchronize();
		Diamond_3 << <ceil((float)N / (block_size*block_size)), (block_size*block_size) >> >(rng, (float*)dev_heightMap, i);
		hipDeviceSynchronize();
	}
	end = clock();

	/* memory copy from device to host*/
	hipMemcpy(heightMap, dev_heightMap, N * sizeof(float), hipMemcpyDeviceToHost);

	/* print the output */
	if (print){
		for (int i = 0; i<N; i++){
			printf("%d: x = %d, y = %d; hm = %f\n", i, i % (SIZE + 1), i / (SIZE + 1), heightMap[i]);
		}
	}
	// printf("\n");	
	// for (int i=0; i<N; i++){
	//   printf("%d: pmi = %f, pmj = %f\n", i, check1[i], check2[i]);
	// }

	// printf("%f\n", cpu_time_used);
	hipFree(dev_heightMap);

	runTime = (double)(end - start) / CLOCKS_PER_SEC;
	printf("Run time for Version_3: %0.20f\n", runTime);
	delete[] heightMap;
	return EXIT_SUCCESS;
}


///////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////// VERSION 4.0 ////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////
/*	VERSION 4.0:
*			4.0 Less Kernel Version: 1 * square kernal + 1 * simple diamond kernel (1 thread => 4 vertex);
*			This kernel combine the four diamond kernel to one single kernel. However, each thread in diamond
*			kernel needs to calculate four vertex.
*/
/* combined diamond kernel to calculate the four point in each thread */
__global__ void Square_Diamond_4(hiprandState* rng, float* hm, int rect){
	/* set idx */
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < N){
		/* initialize vairable */
		int half = rect / 2;
		int i, j, ni, nj, mi, mj;
		int pmi_b, pmj_b, pmi_l, pmj_l, pmi_r, pmj_r, pmi_t, pmj_t;
		float hm_b, hm_l, hm_r, hm_t;
		int num_b, num_l, num_r, num_t;
		int squareInRow = SIZE / rect;

		/* calculate vertex */
		i = (idx%squareInRow*rect) % SIZE;
		j = (idx / squareInRow*rect) % SIZE;
		ni = i + rect;
		nj = j + rect;
		mi = i + half;
		mj = j + half;

		/* set random generator */
		float v1 = (0.0f - (float)ROUGHNESS) / 2;
		float v2 = ((float)ROUGHNESS) / 2;
		hiprandState localState = rng[idx];
		float rand = v1 + (v2 - v1) * hiprand_uniform(&localState);
		rng[idx] = localState;

		/* set height map */
		hm[mi + mj*(SIZE + 1)] = (hm[i + j*(SIZE + 1)] + hm[ni + j*(SIZE + 1)] + hm[i + nj*(SIZE + 1)] + hm[ni + nj*(SIZE + 1)]) / 4 + rand;
		__syncthreads();

		/* find 4 diamond vertex */
		/* bottom vertex */
		pmi_b = mi;
		pmj_b = j;
		/* left vertex */
		pmi_l = i;
		pmj_l = mj;
		/* right vertex */
		pmi_r = ni;
		pmj_r = mj;
		/* top vertex */
		pmi_t = mi;
		pmj_t = nj;

		/* set the value */
		/* bottom height */
		hm_b = 0;
		num_b = 3;
		if (pmj_b - half >= 0){
			hm_b += hm[pmi_b + (pmj_b - half)*(SIZE + 1)];
			num_b = 4;
		}
		hm_b += hm[pmi_b + (pmj_b + half)*(SIZE + 1)];
		hm_b += hm[(pmi_b - half) + pmj_b*(SIZE + 1)];
		hm_b += hm[(pmi_b + half) + pmj_b*(SIZE + 1)];

		/* left height */
		hm_l = 0;
		num_l = 3;
		if (pmi_l - half >= 0){
			hm_l += hm[(pmi_l - half) + pmj_l*(SIZE + 1)];
			num_l = 4;
		}
		hm_l += hm[(pmi_l + half) + pmj_l*(SIZE + 1)];
		hm_l += hm[pmi_l + (pmj_l - half)*(SIZE + 1)];
		hm_l += hm[pmi_l + (pmj_l + half)*(SIZE + 1)];

		/* right height */
		hm_r = 0;
		num_r = 3;
		if (pmi_r + half <= SIZE){
			hm_r += hm[(pmi_r + half) + pmj_r*(SIZE + 1)];
			num_r = 4;
		}
		hm_r += hm[(pmi_r - half) + pmj_r*(SIZE + 1)];
		hm_r += hm[pmi_r + (pmj_r - half)*(SIZE + 1)];
		hm_r += hm[pmi_r + (pmj_r + half)*(SIZE + 1)];

		/* top height */
		hm_t = 0;
		num_t = 3;
		if (pmj_t + half <= SIZE){
			hm_t += hm[pmi_t + (pmj_t + half)*(SIZE + 1)];
			num_t = 4;
		}
		hm_t += hm[pmi_t + (pmj_t - half)*(SIZE + 1)];
		hm_t += hm[(pmi_t - half) + pmj_t*(SIZE + 1)];
		hm_t += hm[(pmi_t + half) + pmj_t*(SIZE + 1)];

		/* get height for  */

		/* set random generator */
		float rand1 = v1 + (v2 - v1) * hiprand_uniform(&localState);
		float rand2 = v1 + (v2 - v1) * hiprand_uniform(&localState);
		float rand3 = v1 + (v2 - v1) * hiprand_uniform(&localState);
		float rand4 = v1 + (v2 - v1) * hiprand_uniform(&localState);

		/* set height map */
		hm[pmi_b + pmj_b*(SIZE + 1)] = hm_b / num_b + rand1;
		hm[pmi_l + pmj_l*(SIZE + 1)] = hm_l / num_l + rand2;
		hm[pmi_r + pmj_r*(SIZE + 1)] = hm_r / num_r + rand3;
		hm[pmi_t + pmj_t*(SIZE + 1)] = hm_t / num_t + rand4;
		// hm[mi + mj*(SIZE+1)] = (hm[i + j*(SIZE+1)] + hm[ni + j*(SIZE+1)] + hm[i + nj*(SIZE+1)] + hm[ni + nj*(SIZE+1)])/4 + rand;
		rng[idx] = localState;
		__syncthreads();
	}
}


/* the host code for version 2: 1 square kernel + 1 stupid diamond kernel */
int version_4(bool print, int block_size) {
	printf("Version 4: Less Kernel Version: 1 * square kernal + 1 * simple diamond kernel (1 thread => 4 vertex)\n");
	/* initialize variables */
	float *heightMap = new float[N];
	/* initialize device */
	float *dev_heightMap;

	/* initialize time*/
	clock_t start, end;
	double runTime;
	/* initial height map */
	for (int i = 0; i<N; i++){
		heightMap[i] = 0;
	}

	/* set height for corner */
	heightMap[0 + 0 * (SIZE + 1)] = CORNER;
	printf("heightMap_corner0: %f\n", heightMap[0 + 0 * (SIZE + 1)]);
	heightMap[SIZE + 0 * (SIZE + 1)] = CORNER;
	printf("heightMap_corner1: %f\n", heightMap[SIZE + 0 * (SIZE + 1)]);
	heightMap[0 + SIZE * (SIZE + 1)] = CORNER;
	printf("heightMap_corner3: %f\n", heightMap[0 + SIZE * (SIZE + 1)]);
	heightMap[SIZE + SIZE * (SIZE + 1)] = CORNER;
	printf("heightMap_corner2: %f\n", heightMap[SIZE + SIZE * (SIZE + 1)]);

	hiprandState* rng;
	/* allocate memory for device */
	hipMalloc(&rng, N * sizeof(hiprandState));
	hipMalloc((void**)&dev_heightMap, N * sizeof(float));

	/* memory copy from host to device */
	hipMemcpy(dev_heightMap, heightMap, N * sizeof(float), hipMemcpyHostToDevice);


	start = clock();
	/* run kernel */
	for (int i = SIZE; i>1; i = i / 2){
		Square_Diamond_4 << <ceil((float)N / 256), 256 >> >(rng, (float*)dev_heightMap, i);
		hipDeviceSynchronize();
	}
	end = clock();

	/* memory copy from device to host*/
	hipMemcpy(heightMap, dev_heightMap, N * sizeof(float), hipMemcpyDeviceToHost);

	/* print the output */
	if (print){
		for (int i = 0; i<N; i++){
			printf("%d: x = %d, y = %d; hm = %f\n", i, i % (SIZE + 1), i / (SIZE + 1), heightMap[i]);
		}
	}
	// printf("%f\n", cpu_time_used);
	hipFree(dev_heightMap);

	runTime = (float)(end - start) / CLOCKS_PER_SEC;
	printf("Run time for Version_4: %0.20f\n", runTime);
	delete[] heightMap;
	return EXIT_SUCCESS;
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////// VERSION 5.0 ////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////
/*	VERSION 5.0:
*			Version 5.0: Kernel Device Version: 1 * kernel + 1 * square device + 1 * diamond device;
*			This version calls two device functions and calls one kernel which loop these two steops
*			within the kernel.
*/

/* smart diamond kernel calculate the diamond vertex and each thread only calculate one vertex */

__device__ void Square_5(hiprandState* rng, float* hm, int rect)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int squareInRow = SIZE / rect;
	if (idx < squareInRow * squareInRow){
		/* initialize vairable */
		int half = rect / 2;
		int i, j, ni, nj, mi, mj;

		/* calculate vertex */
		i = (idx%squareInRow*rect) % SIZE;
		j = (idx / squareInRow*rect) % SIZE;
		ni = i + rect;
		nj = j + rect;
		mi = i + half;
		mj = j + half;


		/* set random generator */
		float v1 = (0.0f - (float)ROUGHNESS) / 2;
		float v2 = ((float)ROUGHNESS) / 2;
		hiprandState localState = rng[idx];
		float rand = v1 + (v2 - v1) * hiprand_uniform(&localState);
		rng[idx] = localState;

		/* set height map */
		hm[mi + mj*(SIZE + 1)] = (hm[i + j*(SIZE + 1)] + hm[ni + j*(SIZE + 1)] + hm[i + nj*(SIZE + 1)] + hm[ni + nj*(SIZE + 1)]) / 4 + rand;
		__syncthreads();
	}
}

__device__ void Diamond_5(hiprandState* rng, float* hm, int rect)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int squareInRow = SIZE / rect;
	if (idx < 4 * squareInRow * squareInRow){
		/* initialize vairable */
		int half = rect / 2;
		int i, j;
		int pmi, pmj;
		float hm_p;
		int num_p;

		/* calculate vertex */
		i = (idx%squareInRow*rect) % SIZE;
		j = (idx / squareInRow*rect) % SIZE;

		/* Calculate the diamond vertex use idx */
		int tid = idx / (squareInRow*squareInRow) % 4;
		pmi = i + (1 - tid % 2)*half + tid / 2 * half;
		pmj = j + tid % 2 * half + tid / 2 * half;

		/* Set the value */
		hm_p = 0;
		num_p = 0;
		if (pmi - half >= 0){
			hm_p += hm[(pmi - half) + pmj*(SIZE + 1)];
			num_p++;
		}
		if (pmi + half <= SIZE){
			hm_p += hm[(pmi + half) + pmj*(SIZE + 1)];
			num_p++;
		}
		if (pmj - half >= 0){
			hm_p += hm[pmi + (pmj - half)*(SIZE + 1)];
			num_p++;
		}
		if (pmj + half <= SIZE){
			hm_p += hm[pmi + (pmj + half)*(SIZE + 1)];
			num_p++;
		}


		/* set random generator */
		float v1 = (0.0f - (float)ROUGHNESS) / 2;
		float v2 = ((float)ROUGHNESS) / 2;
		hiprandState localState = rng[idx];
		float rand = v1 + (v2 - v1) * hiprand_uniform(&localState);

		/* get height for  */
		hm[pmi + pmj*(SIZE + 1)] = hm_p / num_p + rand;
		rng[idx] = localState;
		__syncthreads();
	}
}

__global__ void Square_Diamond_5(hiprandState* rng, float* hm)
{
	for (int i = SIZE; i > 1; i = i / 2)
	{
		__syncthreads();
		Square_5(rng, hm, i);
		Diamond_5(rng, hm, i);
	}
}

int version_5(bool print, int block_size) {
	printf("Version 5.0: Kernel Device Version: 1 * kernel + 1 * square device + 1 * diamond device; \n");
	/* initialize variables */
	float *heightMap = new float[N];
	/* initialize device */
	float *dev_heightMap;
	/* initialize time*/
	clock_t start, end;
	double runTime;
	/* initial height map */
	for (int i = 0; i<N; i++){
		heightMap[i] = 0;
	}

	/* set height for corner */
	heightMap[0 + 0 * (SIZE + 1)] = CORNER;
	printf("heightMap_corner0: %f\n", heightMap[0 + 0 * (SIZE + 1)]);
	heightMap[SIZE + 0 * (SIZE + 1)] = CORNER;
	printf("heightMap_corner1: %f\n", heightMap[SIZE + 0 * (SIZE + 1)]);
	heightMap[0 + SIZE * (SIZE + 1)] = CORNER;
	printf("heightMap_corner3: %f\n", heightMap[0 + SIZE * (SIZE + 1)]);
	heightMap[SIZE + SIZE * (SIZE + 1)] = CORNER;
	printf("heightMap_corner2: %f\n", heightMap[SIZE + SIZE * (SIZE + 1)]);

	hiprandState* rng;
	/* allocate memory for device */
	hipMalloc(&rng, N * sizeof(hiprandState));
	hipMalloc((void**)&dev_heightMap, N * sizeof(float));

	/* memory copy from host to device */
	hipMemcpy(dev_heightMap, heightMap, N * sizeof(float), hipMemcpyHostToDevice);

	start = clock();
	/* run kernel */
	Square_Diamond_5 << <ceil((float)N / 256), 256 >> >(rng, (float*)dev_heightMap);
	hipDeviceSynchronize();
	end = clock();

	/* memory copy from device to host*/
	hipMemcpy(heightMap, dev_heightMap, N * sizeof(float), hipMemcpyDeviceToHost);

	// printf("\n");	
	// for (int i=0; i<N; i++){
	//   printf("%d: pmi = %f, pmj = %f\n", i, check1[i], check2[i]);
	/* print the output */
	if (print){
		for (int i = 0; i<N; i++){
			printf("%d: x = %d, y = %d; hm = %f\n", i, i % (SIZE + 1), i / (SIZE + 1), heightMap[i]);
		}
	}
	// }

	// printf("%f\n", cpu_time_used);
	hipFree(dev_heightMap);

	runTime = (double)(end - start) / CLOCKS_PER_SEC;
	printf("Run time for Version_5: %0.20f\n", runTime);
	delete[] heightMap;
	return EXIT_SUCCESS;
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////// VERSION 6.0 ////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////
/*	VERSION 6.0:
*			6. Less Threads Version: 1 * kernel + 1 * square device + 1 * diamond device (only active threads we need);
*			This kernel combine the four diamond kernel to one single kernel. However, each thread in diamond
*			kernel needs to calculate four vertex.
*/

/* square kernel to calculate the middle point */
__global__ void Square_6(hiprandState* rng, float* hm, int rect){
	/* set idx */
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int squareInRow = SIZE / rect;
	if (idx < squareInRow * squareInRow){
		/* initialize vairable */
		int half = rect / 2;
		int i, j, ni, nj, mi, mj;

		/* calculate vertex */
		i = (idx%squareInRow*rect) % SIZE;
		j = (idx / squareInRow*rect) % SIZE;
		ni = i + rect;
		nj = j + rect;
		mi = i + half;
		mj = j + half;

		/* set check value */
		//	check1[idx] = mi;
		//	check2[idx] = mj;

		/* set random generator */
		float v1 = (0.0f - (float)ROUGHNESS) / 2;
		float v2 = ((float)ROUGHNESS) / 2;
		hiprandState state = rng[idx];
		float random = v1 + (v2 - v1) * (float)hiprand_uniform(&state);
		rng[idx] = state;

		/* set height map */
		hm[mi + mj*(SIZE + 1)] = (hm[i + j*(SIZE + 1)] + hm[ni + j*(SIZE + 1)] + hm[i + nj*(SIZE + 1)] + hm[ni + nj*(SIZE + 1)]) / 4 + random*((float)rect / (float)SIZE);
		__syncthreads();
	}
}

/* smart diamond kernel calculate the diamond vertex and each thread only calculate one vertex */
__global__ void Diamond_6(hiprandState* rng, float* hm, int rect){
	/* set idx */
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int squareInRow = SIZE / rect;
	if (idx < 4 * squareInRow * squareInRow){
		/* initialize vairable */
		int half = rect / 2;
		int i, j;
		int pmi, pmj;
		float hm_p;
		int num_p;

		/* calculate vertex */
		i = (idx%squareInRow*rect) % SIZE;
		j = (idx / squareInRow*rect) % SIZE;

		/* Calculate the diamond vertex use idx */
		int tid = idx / (squareInRow*squareInRow) % 4;
		pmi = i + (1 - tid % 2)*half + tid / 2 * half;
		pmj = j + tid % 2 * half + tid / 2 * half;

		/* Set the value */
		hm_p = 0;
		num_p = 0;
		if (pmi - half >= 0){
			hm_p += hm[(pmi - half) + pmj*(SIZE + 1)];
			num_p++;
		}
		if (pmi + half <= SIZE){
			hm_p += hm[(pmi + half) + pmj*(SIZE + 1)];
			num_p++;
		}
		if (pmj - half >= 0){
			hm_p += hm[pmi + (pmj - half)*(SIZE + 1)];
			num_p++;
		}
		if (pmj + half <= SIZE){
			hm_p += hm[pmi + (pmj + half)*(SIZE + 1)];
			num_p++;
		}

		/* set check value */
		//	check1[idx] = pmi;
		//	check2[idx] = pmj;

		/* set random generator */
		float v1 = (0.0f - (float)ROUGHNESS) / 2;
		float v2 = ((float)ROUGHNESS) / 2;
		hiprandState state = rng[idx];
		float random = v1 + (v2 - v1) * (float)hiprand_uniform(&state);
		rng[idx] = state;

		/* get height for  */
		hm[pmi + pmj*(SIZE + 1)] = hm_p / num_p + random*((float)rect / (float)SIZE);
		__syncthreads();
	}
}

/* the host code for version 3: 1 square kernel + 1 smart diamond kernel */
int version_6(bool print, int block_size) {
	printf("Version 6: square kernel + smart diamond kernel (only active needed threads) \n");
	/* initialize variables */
	float *heightMap = new float[N];
	/* initialize device */
	float *dev_heightMap;
	//	float *dev_check1;
	//	float *dev_check2;
	/* initialize time*/
	clock_t start, end;
	double runTime;
	/* initial height map */
	for (int i = 0; i<N; i++){
		heightMap[i] = 0;
	}

	/* set height for corner */
	heightMap[0 + 0 * (SIZE + 1)] = CORNER;
	printf("heightMap_corner0: %f\n", heightMap[0 + 0 * (SIZE + 1)]);
	heightMap[SIZE + 0 * (SIZE + 1)] = CORNER;
	printf("heightMap_corner1: %f\n", heightMap[SIZE + 0 * (SIZE + 1)]);
	heightMap[0 + SIZE * (SIZE + 1)] = CORNER;
	printf("heightMap_corner3: %f\n", heightMap[0 + SIZE * (SIZE + 1)]);
	heightMap[SIZE + SIZE * (SIZE + 1)] = CORNER;
	printf("heightMap_corner2: %f\n", heightMap[SIZE + SIZE * (SIZE + 1)]);

	hiprandState* rng;
	/* allocate memory for device */
	hipMalloc(&rng, N * sizeof(hiprandState));
	hipMalloc((void**)&dev_heightMap, N * sizeof(float));

	/* memory copy from host to device */
	hipMemcpy(dev_heightMap, heightMap, N * sizeof(float), hipMemcpyHostToDevice);
	//	hipMemcpy(dev_check1, check1, N * sizeof(float), hipMemcpyHostToDevice);
	//	hipMemcpy(dev_check2, check2, N * sizeof(float), hipMemcpyHostToDevice);
	/*
	setseed << < ceil((float)N / 256), 256 >> > (rng, unsigned(time(NULL)));
	float* rand;
	hipMalloc((void**)&rand, N*sizeof(float));
	generate << <ceil((float)N / 256), 256 >> > (rand, rng, N);
	*/
	start = clock();
	/* run kernel */
	for (int i = SIZE; i>1; i = i / 2){
		Square_6 << <ceil((float)N / (block_size*block_size)), block_size*block_size >> >(rng, (float*)dev_heightMap, i);
		hipDeviceSynchronize();
		Diamond_6 << <ceil((float)N / (block_size*block_size)), block_size*block_size >> >(rng, (float*)dev_heightMap, i);
		hipDeviceSynchronize();
	}
	end = clock();

	/* memory copy from device to host*/
	hipMemcpy(heightMap, dev_heightMap, N * sizeof(float), hipMemcpyDeviceToHost);
	//	hipMemcpy(check1, dev_check1, N * sizeof(float), hipMemcpyDeviceToHost);
	//	hipMemcpy(check2, dev_check2, N * sizeof(float), hipMemcpyDeviceToHost);

	/* print the output */
	if (print){
		for (int i = 0; i<N; i++){
			printf("%d: x = %d, y = %d; hm = %f\n", i, i % (SIZE + 1), i / (SIZE + 1), heightMap[i]);
		}
		FILE *fp_out;
		float index_x;
		float index_y;
		fp_out = fopen("vertex.txt", "w");
		for (int j = 0; j<SIZE + 1; j++){
			for (int i = 0; i<SIZE + 1; i++){
				index_x = (float)i / (SIZE / 2) - 1;
				index_y = (float)j / (SIZE / 2) - 1;
				fprintf(fp_out, "%f %f %f\n", index_x, index_y, heightMap[i + j*(SIZE + 1)]);
			}
		}
		fclose(fp_out);
	}
	// printf("\n");	
	// for (int i=0; i<N; i++){
	//   printf("%d: pmi = %f, pmj = %f\n", i, check1[i], check2[i]);
	// }

	// printf("%f\n", cpu_time_used);
	hipFree(dev_heightMap);
	//	hipFree(dev_check1);
	//	hipFree(dev_check2);

	runTime = (double)(end - start) / CLOCKS_PER_SEC;
	printf("Run time for Version_6: %0.20f\n", runTime);
	delete[] heightMap;
	return EXIT_SUCCESS;
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////// VERSION 7.0 ////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////
/*	VERSION 7.0:
*			7. Less Threads Version: 1 * kernel + 1 * square device + 1 * diamond device (only active threads and kernel we need);
*			This kernel combine the four diamond kernel to one single kernel. However, each thread in diamond
*			kernel needs to calculate four vertex.
*/

/* square kernel to calculate the middle point */
__global__ void Square_7(hiprandState* rng, float* hm, int rect){
	/* set idx */
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int squareInRow = SIZE / rect;
	if (idx < squareInRow * squareInRow){
		/* initialize vairable */
		int half = rect / 2;
		int i, j, ni, nj, mi, mj;

		/* calculate vertex */
		i = (idx%squareInRow*rect) % SIZE;
		j = (idx / squareInRow*rect) % SIZE;
		ni = i + rect;
		nj = j + rect;
		mi = i + half;
		mj = j + half;

		/* set random generator */
		float v1 = (0.0f - (float)ROUGHNESS) / 2;
		float v2 = ((float)ROUGHNESS) / 2;
		hiprandState state = rng[idx];
		float random = v1 + (v2 - v1) * (float)hiprand_uniform(&state);
		rng[idx] = state;

		/* set height map */
		hm[mi + mj*(SIZE + 1)] = (hm[i + j*(SIZE + 1)] + hm[ni + j*(SIZE + 1)] + hm[i + nj*(SIZE + 1)] + hm[ni + nj*(SIZE + 1)]) / 4 + random*((float)rect / (float)SIZE);
		__syncthreads();
	}
}

/* smart diamond kernel calculate the diamond vertex and each thread only calculate one vertex */
__global__ void Diamond_7(hiprandState* rng, float* hm, int rect){
	/* set idx */
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int squareInRow = SIZE / rect;
	if (idx < 4 * squareInRow * squareInRow){
		/* initialize vairable */
		int half = rect / 2;
		int i, j;
		int pmi, pmj;
		float hm_p;
		int num_p;

		/* calculate vertex */
		i = (idx%squareInRow*rect) % SIZE;
		j = (idx / squareInRow*rect) % SIZE;

		/* Calculate the diamond vertex use idx */
		int tid = idx / (squareInRow*squareInRow) % 4;
		pmi = i + (1 - tid % 2)*half + tid / 2 * half;
		pmj = j + tid % 2 * half + tid / 2 * half;

		/* Set the value */
		hm_p = 0;
		num_p = 0;
		if (pmi - half >= 0){
			hm_p += hm[(pmi - half) + pmj*(SIZE + 1)];
			num_p++;
		}
		if (pmi + half <= SIZE){
			hm_p += hm[(pmi + half) + pmj*(SIZE + 1)];
			num_p++;
		}
		if (pmj - half >= 0){
			hm_p += hm[pmi + (pmj - half)*(SIZE + 1)];
			num_p++;
		}
		if (pmj + half <= SIZE){
			hm_p += hm[pmi + (pmj + half)*(SIZE + 1)];
			num_p++;
		}


		/* set random generator */
		float v1 = (0.0f - (float)ROUGHNESS) / 2;
		float v2 = ((float)ROUGHNESS) / 2;
		hiprandState state = rng[idx];
		float random = v1 + (v2 - v1) * (float)hiprand_uniform(&state);
		rng[idx] = state;


		/* get height for  */
		hm[pmi + pmj*(SIZE + 1)] = hm_p / num_p + random*((float)rect / (float)SIZE);
		__syncthreads();
	}
}

/* the host code for version 3: 1 square kernel + 1 smart diamond kernel */
int version_7(bool print, int block_size) {
	printf("Version 7: square kernel + smart diamond kernel (only active needed threads&kernel) \n");
	/* initialize variables */
	float *heightMap = new float[N];
	/* initialize device */
	float *dev_heightMap;
	/* initialize time*/
	clock_t start, end;
	double runTime;
	/* initial height map */
	for (int i = 0; i<N; i++){
		heightMap[i] = 0;
	}

	/* set height for corner */
	heightMap[0 + 0 * (SIZE + 1)] = CORNER;
	printf("heightMap_corner0: %f\n", heightMap[0 + 0 * (SIZE + 1)]);
	heightMap[SIZE + 0 * (SIZE + 1)] = CORNER;
	printf("heightMap_corner1: %f\n", heightMap[SIZE + 0 * (SIZE + 1)]);
	heightMap[0 + SIZE * (SIZE + 1)] = CORNER;
	printf("heightMap_corner3: %f\n", heightMap[0 + SIZE * (SIZE + 1)]);
	heightMap[SIZE + SIZE * (SIZE + 1)] = CORNER;
	printf("heightMap_corner2: %f\n", heightMap[SIZE + SIZE * (SIZE + 1)]);

	hiprandState* rng;
	/* allocate memory for device */
	hipMalloc(&rng, N * sizeof(hiprandState));
	hipMalloc((void**)&dev_heightMap, N * sizeof(float));

	/* memory copy from host to device */
	hipMemcpy(dev_heightMap, heightMap, N * sizeof(float), hipMemcpyHostToDevice);

	/*set up random number*/
	/*
	setseed << < ceil((float)N / 256), 256 >> > (rng, unsigned(time(NULL)));
	float* rand;
	hipMalloc((void**)&rand, N*sizeof(float));
	generate << <ceil((float)N / 256), 256 >> > (rand, rng, N);
	*/

	start = clock();
	/* run kernel */
	for (int i = SIZE; i>1; i = i / 2){
		int squareInRow = SIZE / i;
		int size_need = squareInRow * squareInRow;
		int size_need2 = 4 * size_need;
		Square_7 << <ceil((float)size_need / 32), 32 >> >(rng, (float*)dev_heightMap, i);
		hipDeviceSynchronize();
		Diamond_7 << <ceil((float)size_need2 / 32), 32 >> >(rng, (float*)dev_heightMap, i);
		hipDeviceSynchronize();
	}
	end = clock();

	/* memory copy from device to host*/
	hipMemcpy(heightMap, dev_heightMap, N * sizeof(float), hipMemcpyDeviceToHost);

	/* print the output */
	if (print){
		for (int i = 0; i<N; i++){
			printf("%d: x = %d, y = %d; hm = %f\n", i, i % (SIZE + 1), i / (SIZE + 1), heightMap[i]);
		}
		FILE *fp_out;
		float index_x;
		float index_y;
		fp_out = fopen("vertex.txt", "w");
		for (int j = 0; j<SIZE + 1; j++){
			for (int i = 0; i<SIZE + 1; i++){
				index_x = (float)i / (SIZE / 2) - 1;
				index_y = (float)j / (SIZE / 2) - 1;
				fprintf(fp_out, "%f %f %f\n", index_x, index_y, heightMap[i + j*(SIZE + 1)]);
			}
		}
		fclose(fp_out);
	}
	// printf("\n");	
	// for (int i=0; i<N; i++){
	//   printf("%d: pmi = %f, pmj = %f\n", i, check1[i], check2[i]);
	// }

	// printf("%f\n", cpu_time_used);
	hipFree(dev_heightMap);

	runTime = (double)(end - start) / CLOCKS_PER_SEC;
	printf("Run time for Version_7: %0.20f\n", runTime);
	delete[] heightMap;
	return EXIT_SUCCESS;
}


int version_8(void) {
	printf("8\n");
	return 0;
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////// VERSION 9.0 ////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////
/*	VERSION 9.0:
*			9.0 Smarter Kernel Version: 1 * sqaure kernel + 1 * smart diamond kernel (1 thread => 1 vertex);
*			This version reconstruct the diamond kernel to use different threads for different vertx. Each
*			thread in diamond kernel only need to calculate one vertex. (A simple revised 2D version of version 3)
*/
__global__ void Square_9(hiprandState* rng, float* hm, int rect){
	/* set idx */
	int idx_temp = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	if (idx_temp < SIZE + 1 && idy < SIZE + 1){
		int idx = idy*(SIZE + 1) + idx_temp;
		/* initialize vairable */
		int half = rect / 2;
		int i, j, ni, nj, mi, mj;
		int squareInRow = SIZE / rect;

		/* calculate vertex */
		i = (idx%squareInRow*rect) % SIZE;
		j = (idx / squareInRow*rect) % SIZE;
		ni = i + rect;
		nj = j + rect;
		mi = i + half;
		mj = j + half;

		/* set random generator */
		float v1 = (0.0f - (float)ROUGHNESS) / 2;
		float v2 = ((float)ROUGHNESS) / 2;
		hiprandState state = rng[idx];
		float random = v1 + (v2 - v1) * (float)hiprand_uniform(&state);
		rng[idx] = state;

		/* set height map */
		hm[mi + mj*(SIZE + 1)] = (hm[i + j*(SIZE + 1)] + hm[ni + j*(SIZE + 1)] + hm[i + nj*(SIZE + 1)] + hm[ni + nj*(SIZE + 1)]) / 4 + random*((float)rect / (float)SIZE);
		__syncthreads();
	}
}

__global__ void Diamond_9(hiprandState* rng, float* hm, int rect){
	/* set idx */
	int idx_temp = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	if (idx_temp < SIZE + 1 && idy < SIZE + 1){
		int idx = idy*(SIZE + 1) + idx_temp;
		/* initialize vairable */
		int half = rect / 2;
		int i, j;
		int pmi, pmj;
		float hm_p;
		int num_p;
		int squareInRow = SIZE / rect;

		/* calculate vertex */
		i = (idx%squareInRow*rect) % SIZE;
		j = (idx / squareInRow*rect) % SIZE;

		/* Calculate the diamond vertex use idx */
		int tid = idx / (squareInRow*squareInRow) % 4;
		pmi = i + (1 - tid % 2)*half + tid / 2 * half;
		pmj = j + tid % 2 * half + tid / 2 * half;

		/* Set the value */
		hm_p = 0;
		num_p = 0;
		if (pmi - half >= 0){
			hm_p += hm[(pmi - half) + pmj*(SIZE + 1)];
			num_p++;
		}
		if (pmi + half <= SIZE){
			hm_p += hm[(pmi + half) + pmj*(SIZE + 1)];
			num_p++;
		}
		if (pmj - half >= 0){
			hm_p += hm[pmi + (pmj - half)*(SIZE + 1)];
			num_p++;
		}
		if (pmj + half <= SIZE){
			hm_p += hm[pmi + (pmj + half)*(SIZE + 1)];
			num_p++;
		}

		/* set random generator */
		float v1 = (0.0f - (float)ROUGHNESS) / 2;
		float v2 = ((float)ROUGHNESS) / 2;
		hiprandState state = rng[idx];
		float random = v1 + (v2 - v1) * (float)hiprand_uniform(&state);
		rng[idx] = state;


		/* get height for  */
		hm[pmi + pmj*(SIZE + 1)] = hm_p / num_p + random*((float)rect / (float)SIZE);
		__syncthreads();
	}
}

/* the host code for version 8: 2D + 1 square kernel + 1 smart diamond kernel. */
int version_9(bool print, int block_size) {
	printf("Version 9: 1 * sqaure kernel + 1 * smart diamond kernel (1 thread => 1 vertex)\n");
	/* initialize variables */
	float *heightMap = new float[N];
	/* initialize device */
	float *dev_heightMap;
	/* initialize time*/
	clock_t start, end;
	double runTime;
	/* initial height map */
	for (int i = 0; i<N; i++){
		heightMap[i] = 0;
	}

	/* set height for corner */
	heightMap[0 + 0 * (SIZE + 1)] = 1;
	printf("heightMap_corner0: %f\n", heightMap[0 + 0 * (SIZE + 1)]);
	heightMap[SIZE + 0 * (SIZE + 1)] = 2;
	printf("heightMap_corner1: %f\n", heightMap[SIZE + 0 * (SIZE + 1)]);
	heightMap[0 + SIZE * (SIZE + 1)] = 3;
	printf("heightMap_corner3: %f\n", heightMap[0 + SIZE * (SIZE + 1)]);
	heightMap[SIZE + SIZE * (SIZE + 1)] = 4;
	printf("heightMap_corner2: %f\n", heightMap[SIZE + SIZE * (SIZE + 1)]);

	hiprandState* rng;
	/* allocate memory for device */
	hipMalloc(&rng, N * sizeof(hiprandState));
	hipMalloc((void**)&dev_heightMap, N * sizeof(float));

	/* memory copy from host to device */
	hipMemcpy(dev_heightMap, heightMap, N * sizeof(float), hipMemcpyHostToDevice);
	
	/*set up random number*/
	/*
	setseed << < ceil((float)N / (block_size*block_size)), (block_size*block_size) >> > (rng, unsigned(time(NULL)));
	float* rand;
	hipMalloc((void**)&rand, N*sizeof(float));
	generate << <ceil((float)N / (block_size*block_size)), (block_size*block_size) >> > (rand, rng, N);
	*/
	start = clock();
	/* run kernel */
	dim3 DimGrid(ceil(((float)SIZE) / block_size), ceil(((float)SIZE) / block_size), 1);
	dim3 DimBlock(block_size, block_size, 1);
	for (int i = SIZE; i>1; i = i / 2){
		Square_9 << <DimGrid, DimBlock >> >(rng, (float*)dev_heightMap, i);
		hipDeviceSynchronize();
		Diamond_9 << <DimGrid, DimBlock >> >(rng, (float*)dev_heightMap, i);
		hipDeviceSynchronize();
	}
	end = clock();

	/* memory copy from device to host*/
	hipMemcpy(heightMap, dev_heightMap, N * sizeof(float), hipMemcpyDeviceToHost);

	/* print the output */
	if (print){
		for (int i = 0; i<N; i++){
			printf("%d: x = %d, y = %d; hm = %f\n", i, i % (SIZE + 1), i / (SIZE + 1), heightMap[i]);
		}
		FILE *fp_out;
		float index_x;
		float index_y;
		fp_out = fopen("vertex.txt", "w");
		for (int j = 0; j<SIZE + 1; j++){
			for (int i = 0; i<SIZE + 1; i++){
				index_x = (float)i / (SIZE / 2) - 1;
				index_y = (float)j / (SIZE / 2) - 1;
				fprintf(fp_out, "%f %f %f\n", index_x, index_y, heightMap[i + j*(SIZE + 1)]);
			}
		}
		fclose(fp_out);
	}
	// printf("\n");	
	//	for (int i=0; i<SIZE+1; i++){
	//	   printf("%d: pmi = %f, pmj = %f\n", i, check1[i], check2[i]);
	//	}

	// printf("%f\n", cpu_time_used);
	hipFree(dev_heightMap);

	runTime = (double)(end - start) / CLOCKS_PER_SEC;
	printf("Run time for Version_9: %0.20f\n", runTime);
	delete[] heightMap;
	return EXIT_SUCCESS;
}


///////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////// VERSION 10.0 ////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////
/*	VERSION 10.0:
*			10.0 Smarter Kernel Version: 1 * sqaure kernel + 1 * smart diamond kernel (1 thread => 1 vertex);
*			This version reconstruct the diamond kernel to use different threads for different vertx. Each
*			thread in diamond kernel only need to calculate one vertex. (A smart revised 2D version of version 3)
*/
__global__ void Square_10(hiprandState* rng, float* hm, int rect){
	/* set idx */
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int squareInRow = SIZE / rect;
	if (idx < squareInRow && idy < squareInRow){
		/* initialize vairable */
		int half = rect / 2;
		int i, j, ni, nj, mi, mj;

		/* calculate vertex */
		i = idx;
		j = idy;
		ni = i + rect;
		nj = j + rect;
		mi = i + half;
		mj = j + half;

		/* set random generator */
		float v1 = (0.0f - (float)ROUGHNESS) / 2;
		float v2 = ((float)ROUGHNESS) / 2;
		hiprandState state = rng[idx];
		float random = v1 + (v2 - v1) * (float)hiprand_uniform(&state);
		rng[idx] = state;

		/* set height map */
		hm[mi + mj*(SIZE + 1)] = (hm[i + j*(SIZE + 1)] + hm[ni + j*(SIZE + 1)] + hm[i + nj*(SIZE + 1)] + hm[ni + nj*(SIZE + 1)]) / 4 + random*((float)rect / (float)SIZE);
		__syncthreads();
	}
}

__global__ void Diamond_10(hiprandState* rng, float* hm, int rect){
	/* set idx */
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int squareInRow = SIZE / rect;
	if (idx < 2*squareInRow && idy < 2*squareInRow){
		/* initialize vairable */
		int half = rect / 2;
		int i, j;
		int pmi, pmj;
		float hm_p;
		int num_p;

		/* calculate vertex */
		i = idx;
		j = idy;

		/* Calculate the diamond vertex use idx */
		int tid = idx / (squareInRow) % 4;
		pmi = i + (1 - tid % 2)*half + tid / 2 * half;
		pmj = j + tid % 2 * half + tid / 2 * half;

		/* Set the value */
		hm_p = 0;
		num_p = 0;
		if (pmi - half >= 0){
			hm_p += hm[(pmi - half) + pmj*(SIZE + 1)];
			num_p++;
		}
		if (pmi + half <= SIZE){
			hm_p += hm[(pmi + half) + pmj*(SIZE + 1)];
			num_p++;
		}
		if (pmj - half >= 0){
			hm_p += hm[pmi + (pmj - half)*(SIZE + 1)];
			num_p++;
		}
		if (pmj + half <= SIZE){
			hm_p += hm[pmi + (pmj + half)*(SIZE + 1)];
			num_p++;
		}

		/* set random generator */
		float v1 = (0.0f - (float)ROUGHNESS) / 2;
		float v2 = ((float)ROUGHNESS) / 2;
		hiprandState state = rng[idx];
		float random = v1 + (v2 - v1) * (float)hiprand_uniform(&state);
		rng[idx] = state;


		/* get height for  */
		hm[pmi + pmj*(SIZE + 1)] = hm_p / num_p + random*((float)rect / (float)SIZE);
		__syncthreads();
	}
}

/* the host code for version 10: 2D(more smart) + 1 square kernel + 1 smart diamond kernel. */
int version_10(bool print, int block_size) {
	printf("Version 10: square kernel + smart diamond kernel\n");
	/* initialize variables */
	float *heightMap = new float[N];
	/* initialize device */
	float *dev_heightMap;
	/* initialize time*/
	clock_t start, end;
	double runTime;
	/* initial height map */
	for (int i = 0; i<N; i++){
		heightMap[i] = 0;
	}

	/* set height for corner */
	heightMap[0 + 0 * (SIZE + 1)] = 1;
	printf("heightMap_corner0: %f\n", heightMap[0 + 0 * (SIZE + 1)]);
	heightMap[SIZE + 0 * (SIZE + 1)] = 2;
	printf("heightMap_corner1: %f\n", heightMap[SIZE + 0 * (SIZE + 1)]);
	heightMap[0 + SIZE * (SIZE + 1)] = 3;
	printf("heightMap_corner3: %f\n", heightMap[0 + SIZE * (SIZE + 1)]);
	heightMap[SIZE + SIZE * (SIZE + 1)] = 4;
	printf("heightMap_corner2: %f\n", heightMap[SIZE + SIZE * (SIZE + 1)]);

	hiprandState* rng;
	/* allocate memory for device */
	hipMalloc(&rng, N * sizeof(hiprandState));
	hipMalloc((void**)&dev_heightMap, N * sizeof(float));

	/* memory copy from host to device */
	hipMemcpy(dev_heightMap, heightMap, N * sizeof(float), hipMemcpyHostToDevice);

	/*set up random number*/
	/*
	setseed << < ceil((float)N / (block_size*block_size)), (block_size*block_size) >> > (rng, unsigned(time(NULL)));
	float* rand;
	hipMalloc((void**)&rand, N*sizeof(float));
	generate << <ceil((float)N / (block_size*block_size)), (block_size*block_size) >> > (rand, rng, N);
	*/
	/* run kernel */
	dim3 DimGrid(ceil(((float)SIZE) / block_size), ceil(((float)SIZE) / block_size), 1);
	dim3 DimBlock(block_size, block_size, 1);
	start = clock();
	for (int i = SIZE; i>1; i = i / 2){
		Square_10 << <DimGrid, DimBlock >> >(rng, (float*)dev_heightMap, i);
		hipDeviceSynchronize();
		Diamond_10 << <DimGrid, DimBlock >> >(rng, (float*)dev_heightMap, i);
		hipDeviceSynchronize();
	}
	end = clock();

	/* memory copy from device to host*/
	hipMemcpy(heightMap, dev_heightMap, N * sizeof(float), hipMemcpyDeviceToHost);

	/* print the output */
	if (print){
		for (int i = 0; i<N; i++){
			printf("%d: x = %d, y = %d; hm = %f\n", i, i % (SIZE + 1), i / (SIZE + 1), heightMap[i]);
		}
		FILE *fp_out;
		float index_x;
		float index_y;
		fp_out = fopen("vertex.txt", "w");
		for (int j = 0; j<SIZE + 1; j++){
			for (int i = 0; i<SIZE + 1; i++){
				index_x = (float)i / (SIZE / 2) - 1;
				index_y = (float)j / (SIZE / 2) - 1;
				fprintf(fp_out, "%f %f %f\n", index_x, index_y, heightMap[i + j*(SIZE + 1)]);
			}
		}
		fclose(fp_out);
	}
	// printf("\n");	
	//	for (int i=0; i<SIZE+1; i++){
	//	   printf("%d: pmi = %f, pmj = %f\n", i, check1[i], check2[i]);
	//	}

	// printf("%f\n", cpu_time_used);
	hipFree(dev_heightMap);


	runTime = (double)(end - start) / CLOCKS_PER_SEC;
	printf("Run time for Version_10: %0.20f\n", runTime);
	delete[] heightMap;
	return EXIT_SUCCESS;
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////// VERSION 11.0 ////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////
/*	VERSION 11.0:
*			11.0 Smarter Kernel Version: 1 * sqaure kernel + 1 * smart diamond kernel (1 thread => 1 vertex);
*			This version reconstruct the diamond kernel to use different threads for different vertx. Each
*			thread in diamond kernel only need to calculate one vertex. (A smart revised 2D version of version 6)
*/
__global__ void Square_11(hiprandState* rng, float** hm, int rect){
	/* set idx */
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int squareInRow = SIZE / rect;
	if (idx < squareInRow && idy < squareInRow){
		/* initialize vairable */
		int half = rect / 2;
		int i, j, ni, nj, mi, mj;

		/* calculate vertex */
		i = idx;
		j = idy;
		ni = i + rect;
		nj = j + rect;
		mi = i + half;
		mj = j + half;

		/* set random generator */
		float v1 = (0.0f - (float)ROUGHNESS) / 2;
		float v2 = ((float)ROUGHNESS) / 2;
		hiprandState localState = rng[idx];
		float rand = v1 + (v2 - v1) * hiprand_uniform(&localState);
		rng[idx] = localState;

		/* set height map */
		hm[mi][mj] = (hm[i ][j] + hm[ni][j] + hm[i][nj] + hm[ni][nj]) / 4 + rand;
		__syncthreads();
	}
}

__global__ void Diamond_11(hiprandState* rng, float** hm, int rect){
	/* set idx */
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int squareInRow = SIZE / rect;
	if (idx < 2 * squareInRow && idy < 2 * squareInRow){
		/* initialize vairable */
		int half = rect / 2;
		int i, j;
		int pmi, pmj;
		float hm_p;
		int num_p;
		int squareInRow = SIZE / rect;

		/* calculate vertex */
		i = idx;
		j = idy;

		/* Calculate the diamond vertex use idx */
		int tid = idx / (squareInRow) % 4;
		pmi = i + (1 - tid % 2)*half + tid / 2 * half;
		pmj = j + tid % 2 * half + tid / 2 * half;

		/* Set the value */
		hm_p = 0;
		num_p = 0;
		if (pmi - half >= 0){
			hm_p += hm[(pmi - half)][pmj];
			num_p++;
		}
		if (pmi + half <= SIZE){
			hm_p += hm[(pmi + half)][pmj];
			num_p++;
		}
		if (pmj - half >= 0){
			hm_p += hm[pmi] [(pmj - half)];
			num_p++;
		}
		if (pmj + half <= SIZE){
			hm_p += hm[pmi][(pmj + half)];
			num_p++;
		}

		/* set random generator */
		float v1 = (0.0f - (float)ROUGHNESS) / 2;
		float v2 = ((float)ROUGHNESS) / 2;
		hiprandState localState = rng[idx];
		float rand = v1 + (v2 - v1) * hiprand_uniform(&localState);

		/* get height for  */
		hm[pmi][pmj] = hm_p / num_p + rand;
		rng[idx] = localState;
		__syncthreads();
	}
}

/* the host code for version 1: 2D(more smart) + 1 square kernel + 1 smart diamond kernel. */
int version_11(bool print, int block_size) {
	printf("Version 11: square kernel + smart diamond kernel\n");
	/* initialize variables */
	float **heightMap = new float*[SIZE + 1];
	/* initialize device */
	float *dev_heightMap;
	/* initialize time*/
	clock_t start, end;
	double runTime;
	/* initial height map */
	for (int i = 0; i<SIZE + 1; i++){
		heightMap[i] = new float[SIZE + 1];
		for (int j = 0; j<SIZE + 1; j++){
			heightMap[i][j] = 0.0;
		}
	}

	/* set height for corner */
	heightMap[0][0] = 1;
	printf("heightMap_corner0: %f\n", heightMap[0][0]);
	heightMap[SIZE][0] = 2;
	printf("heightMap_corner1: %f\n", heightMap[SIZE][0]);
	heightMap[0][SIZE] = 3;
	printf("heightMap_corner3: %f\n", heightMap[0][SIZE]);
	heightMap[SIZE][SIZE] = 4;
	printf("heightMap_corner2: %f\n", heightMap[SIZE][SIZE]);

	hiprandState* rng;
	/* allocate memory for device */
	hipMalloc(&rng, N * sizeof(hiprandState));
	hipMalloc((void**)&dev_heightMap, N * sizeof(float));

	/* memory copy from host to device */
	hipMemcpy(dev_heightMap, heightMap, N * sizeof(float), hipMemcpyHostToDevice);

	start = clock();
	/* run kernel */
	dim3 DimGrid(ceil(((float)SIZE) / block_size), ceil(((float)SIZE) / block_size), 1);
	dim3 DimBlock(block_size, block_size, 1);
	for (int i = SIZE; i>1; i = i / 2){
		Square_11 << <DimGrid, DimBlock >> >(rng, (float**)dev_heightMap, i);
		hipDeviceSynchronize();
		Diamond_11 << <DimGrid, DimBlock >> >(rng, (float**)dev_heightMap, i);
		hipDeviceSynchronize();
	}
	end = clock();

	/* memory copy from device to host*/
	hipMemcpy(heightMap, dev_heightMap, N * sizeof(float), hipMemcpyDeviceToHost);

	/* print the output */
//	if (print){
//		for (int i = 0; i<N; i++){
//			printf("%d: x = %d, y = %d; hm = %f\n", i, i % (SIZE + 1), i / (SIZE + 1), heightMap[i]);
//		}
//	}
	// printf("\n");	
	//	for (int i=0; i<SIZE+1; i++){
	//	   printf("%d: pmi = %f, pmj = %f\n", i, check1[i], check2[i]);
	//	}

	// printf("%f\n", cpu_time_used);
	hipFree(dev_heightMap);


	runTime = (double)(end - start) / CLOCKS_PER_SEC;
	printf("Run time for Version_11: %0.20f\n", runTime);

	for (int i = 0; i<SIZE + 1; i++)
		delete[] heightMap[i];
	delete[] heightMap;
	return EXIT_SUCCESS;
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////// VERSION 12.0 ////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////
/*	VERSION 12.0:
*			12.0 Smarter Kernel Version: 1 * sqaure kernel + 1 * smart diamond kernel (1 thread => 1 vertex);
*			This version reconstruct the diamond kernel to use different threads for different vertx. Each
*			thread in diamond kernel only need to calculate one vertex. (A smart revised 2D version of version 3)
*/
__global__ void Square_12(hiprandState* rng, float* hm, int rect){
	/* set idx */
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int squareInRow = SIZE / rect;
	if (idx < squareInRow && idy < squareInRow){
		/* initialize vairable */
		int half = rect / 2;
		int i, j, ni, nj, mi, mj;

		/* calculate vertex */
		i = idx;
		j = idy;
		ni = i + rect;
		nj = j + rect;
		mi = i + half;
		mj = j + half;

		/* set random generator */
		float v1 = (0.0f - (float)ROUGHNESS) / 2;
		float v2 = ((float)ROUGHNESS) / 2;
		hiprandState state = rng[idx];
		float random = v1 + (v2 - v1) * (float)hiprand_uniform(&state);
		rng[idx] = state;

		/* set height map */
		hm[mi + mj*(SIZE + 1)] = (hm[i + j*(SIZE + 1)] + hm[ni + j*(SIZE + 1)] + hm[i + nj*(SIZE + 1)] + hm[ni + nj*(SIZE + 1)]) / 4 + random*((float)rect / (float)SIZE);
		__syncthreads();
	}
}

__global__ void Diamond_12(hiprandState* rng, float* hm, int rect){
	/* set idx */
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int squareInRow = SIZE / rect;
	if (idx < 2 * squareInRow && idy < 2 * squareInRow){
		/* initialize vairable */
		int half = rect / 2;
		int i, j;
		int pmi, pmj;
		float hm_p;
		int num_p;

		/* calculate vertex */
		i = idx;
		j = idy;

		/* Calculate the diamond vertex use idx */
		int tid = idx / (squareInRow) % 4;
		pmi = i + (1 - tid % 2)*half + tid / 2 * half;
		pmj = j + tid % 2 * half + tid / 2 * half;

		/* Set the value */
		hm_p = 0;
		num_p = 0;
		if (pmi - half >= 0){
			hm_p += hm[(pmi - half) + pmj*(SIZE + 1)];
			num_p++;
		}
		if (pmi + half <= SIZE){
			hm_p += hm[(pmi + half) + pmj*(SIZE + 1)];
			num_p++;
		}
		if (pmj - half >= 0){
			hm_p += hm[pmi + (pmj - half)*(SIZE + 1)];
			num_p++;
		}
		if (pmj + half <= SIZE){
			hm_p += hm[pmi + (pmj + half)*(SIZE + 1)];
			num_p++;
		}

		/* set random generator */
		float v1 = (0.0f - (float)ROUGHNESS) / 2;
		float v2 = ((float)ROUGHNESS) / 2;
		hiprandState state = rng[idx];
		float random = v1 + (v2 - v1) * (float)hiprand_uniform(&state);
		rng[idx] = state;


		/* get height for  */
		hm[pmi + pmj*(SIZE + 1)] = hm_p / num_p + random*((float)rect / (float)SIZE);
		__syncthreads();
	}
}

/* the host code for version 12: 2D(more smart) + 1 square kernel + 1 smart diamond kernel. */
int version_12(bool print, int block_size) {
	printf("Version 12: square kernel + smart diamond kernel\n");
	/* initialize variables */
	float *heightMap = new float[N];
	/* initialize device */
	float *dev_heightMap;
	/* initialize time*/
	clock_t start, end;
	double runTime;
	/* initial height map */
	for (int i = 0; i<N; i++){
		heightMap[i] = 0;
	}

	/* set height for corner */
	heightMap[0 + 0 * (SIZE + 1)] = 1;
	printf("heightMap_corner0: %f\n", heightMap[0 + 0 * (SIZE + 1)]);
	heightMap[SIZE + 0 * (SIZE + 1)] = 2;
	printf("heightMap_corner1: %f\n", heightMap[SIZE + 0 * (SIZE + 1)]);
	heightMap[0 + SIZE * (SIZE + 1)] = 3;
	printf("heightMap_corner3: %f\n", heightMap[0 + SIZE * (SIZE + 1)]);
	heightMap[SIZE + SIZE * (SIZE + 1)] = 4;
	printf("heightMap_corner2: %f\n", heightMap[SIZE + SIZE * (SIZE + 1)]);

	hiprandState* rng;
	/* allocate memory for device */
	hipMalloc(&rng, N * sizeof(hiprandState));
	hipMalloc((void**)&dev_heightMap, N * sizeof(float));

	/* memory copy from host to device */
	hipMemcpy(dev_heightMap, heightMap, N * sizeof(float), hipMemcpyHostToDevice);

	/*set up random number*/
	/*
	setseed << < ceil((float)N / (block_size*block_size)), (block_size*block_size) >> > (rng, unsigned(time(NULL)));
	float* rand;
	hipMalloc((void**)&rand, N*sizeof(float));
	generate << <ceil((float)N / (block_size*block_size)), (block_size*block_size) >> > (rand, rng, N);
	*/
	/* run kernel */
	
	start = clock();
	for (int i = SIZE; i>1; i = i / 2){
		int size_need = SIZE / i;
		dim3 DimGrid(ceil(((float)size_need) / block_size), ceil(((float)size_need) / block_size), 1);
		dim3 DimBlock(block_size, block_size, 1);
		Square_12 << <DimGrid, DimBlock >> >(rng, (float*)dev_heightMap, i);
		hipDeviceSynchronize();
		Diamond_12 << <DimGrid, DimBlock >> >(rng, (float*)dev_heightMap, i);
		hipDeviceSynchronize();
	}
	end = clock();

	/* memory copy from device to host*/
	hipMemcpy(heightMap, dev_heightMap, N * sizeof(float), hipMemcpyDeviceToHost);

	/* print the output */
	if (print){
		for (int i = 0; i<N; i++){
			printf("%d: x = %d, y = %d; hm = %f\n", i, i % (SIZE + 1), i / (SIZE + 1), heightMap[i]);
		}
		FILE *fp_out;
		float index_x;
		float index_y;
		fp_out = fopen("vertex.txt", "w");
		for (int j = 0; j<SIZE + 1; j++){
			for (int i = 0; i<SIZE + 1; i++){
				index_x = (float)i / (SIZE / 2) - 1;
				index_y = (float)j / (SIZE / 2) - 1;
				fprintf(fp_out, "%f %f %f\n", index_x, index_y, heightMap[i + j*(SIZE + 1)]);
			}
		}
		fclose(fp_out);
	}
	// printf("\n");	
	//	for (int i=0; i<SIZE+1; i++){
	//	   printf("%d: pmi = %f, pmj = %f\n", i, check1[i], check2[i]);
	//	}

	// printf("%f\n", cpu_time_used);
	hipFree(dev_heightMap);


	runTime = (double)(end - start) / CLOCKS_PER_SEC;
	printf("Run time for Version_12: %0.20f\n", runTime);
	delete[] heightMap;
	return EXIT_SUCCESS;
}


///////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////// VERSION 100.0 ////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////
/*	VERSION 100.0:
*			100.0 Less Kernel Version: 1 * square kernal and simple diamond kernel (1 thread => 4 vertex);
*			This kernel combine the four diamond kernel to one single kernel. However, each thread in diamond
*			kernel needs to calculate four vertex. (from version 4)
*/
/* combined diamond kernel to calculate the four point in each thread */
__global__ void Square_Diamond_100(hiprandState* rng, float* hm, int rect, float* check1, float* check2){
	/* set idx */
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ float subTile[256];
	subTile[idx % 256] = hm[idx];
	for (int rect = SIZE; rect>1; rect = rect / 2){
		if (idx < N){
			/* initialize vairable */
			int half = rect / 2;
			int i, j, ni, nj, mi, mj;
			int pmi_b, pmj_b, pmi_l, pmj_l, pmi_r, pmj_r, pmi_t, pmj_t;
			float hm_b, hm_l, hm_r, hm_t;
			int num_b, num_l, num_r, num_t;
			int squareInRow = SIZE / rect;

			/* calculate vertex */
			i = (idx%squareInRow*rect) % SIZE;
			j = (idx / squareInRow*rect) % SIZE;
			ni = i + rect;
			nj = j + rect;
			mi = i + half;
			mj = j + half;

			/* set check value */
			check1[idx] = mi;
			check2[idx] = mj;

			/* set random generator */
			float v1 = (0.0f - (float)ROUGHNESS) / 2;
			float v2 = ((float)ROUGHNESS) / 2;
			hiprandState localState = rng[idx];
			float rand = v1 + (v2 - v1) * hiprand_uniform(&localState);
			rng[idx] = localState;

			/* set height map */
			hm[mi + mj*(SIZE + 1)] = (hm[i + j*(SIZE + 1)] + hm[ni + j*(SIZE + 1)] + hm[i + nj*(SIZE + 1)] + hm[ni + nj*(SIZE + 1)]) / 4 + rand;
			__syncthreads();

			/* find 4 diamond vertex */
			/* bottom vertex */
			pmi_b = mi;
			pmj_b = j;
			/* left vertex */
			pmi_l = i;
			pmj_l = mj;
			/* right vertex */
			pmi_r = ni;
			pmj_r = mj;
			/* top vertex */
			pmi_t = mi;
			pmj_t = nj;

			/* set the value */
			/* bottom height */
			hm_b = 0;
			num_b = 3;
			if (pmj_b - half >= 0){
				hm_b += hm[pmi_b + (pmj_b - half)*(SIZE + 1)];
				num_b = 4;
			}
			hm_b += hm[pmi_b + (pmj_b + half)*(SIZE + 1)];
			hm_b += hm[(pmi_b - half) + pmj_b*(SIZE + 1)];
			hm_b += hm[(pmi_b + half) + pmj_b*(SIZE + 1)];

			/* left height */
			hm_l = 0;
			num_l = 3;
			if (pmi_l - half >= 0){
				hm_l += hm[(pmi_l - half) + pmj_l*(SIZE + 1)];
				num_l = 4;
			}
			hm_l += hm[(pmi_l + half) + pmj_l*(SIZE + 1)];
			hm_l += hm[pmi_l + (pmj_l - half)*(SIZE + 1)];
			hm_l += hm[pmi_l + (pmj_l + half)*(SIZE + 1)];

			/* right height */
			hm_r = 0;
			num_r = 3;
			if (pmi_r + half <= SIZE){
				hm_r += hm[(pmi_r + half) + pmj_r*(SIZE + 1)];
				num_r = 4;
			}
			hm_r += hm[(pmi_r - half) + pmj_r*(SIZE + 1)];
			hm_r += hm[pmi_r + (pmj_r - half)*(SIZE + 1)];
			hm_r += hm[pmi_r + (pmj_r + half)*(SIZE + 1)];

			/* top height */
			hm_t = 0;
			num_t = 3;
			if (pmj_t + half <= SIZE){
				hm_t += hm[pmi_t + (pmj_t + half)*(SIZE + 1)];
				num_t = 4;
			}
			hm_t += hm[pmi_t + (pmj_t - half)*(SIZE + 1)];
			hm_t += hm[(pmi_t - half) + pmj_t*(SIZE + 1)];
			hm_t += hm[(pmi_t + half) + pmj_t*(SIZE + 1)];

			/* set check value */
			check1[idx] = hm_l;
			check2[idx] = hm_l;

			/* get height for  */

			/* set random generator */
			float rand1 = v1 + (v2 - v1) * hiprand_uniform(&localState);
			float rand2 = v1 + (v2 - v1) * hiprand_uniform(&localState);
			float rand3 = v1 + (v2 - v1) * hiprand_uniform(&localState);
			float rand4 = v1 + (v2 - v1) * hiprand_uniform(&localState);

			/* set height map */
			hm[pmi_b + pmj_b*(SIZE + 1)] = hm_b / num_b + rand1;
			hm[pmi_l + pmj_l*(SIZE + 1)] = hm_l / num_l + rand2;
			hm[pmi_r + pmj_r*(SIZE + 1)] = hm_r / num_r + rand3;
			hm[pmi_t + pmj_t*(SIZE + 1)] = hm_t / num_t + rand4;
			// hm[mi + mj*(SIZE+1)] = (hm[i + j*(SIZE+1)] + hm[ni + j*(SIZE+1)] + hm[i + nj*(SIZE+1)] + hm[ni + nj*(SIZE+1)])/4 + rand;
			rng[idx] = localState;
			__syncthreads();
		}
	}
}


/* the host code for version 2: 1 square kernel + 1 stupid diamond kernel */
int version_100(bool print) {
	printf("Version 4: Less Kernel Version: 1 * square kernal + 1 * simple diamond kernel (1 thread => 4 vertex)\n");
	/* initialize variables */
	float check1[N];
	float check2[N];
	float heightMap[N];
	/* initialize device */
	float *dev_heightMap;
	float *dev_check1;
	float *dev_check2;
	/* initialize time*/
	clock_t start, end;
	double runTime;
	/* initial height map */
	for (int i = 0; i<N; i++){
		heightMap[i] = 0;
	}

	/* set height for corner */
	heightMap[0 + 0 * (SIZE + 1)] = CORNER;
	printf("heightMap_corner0: %f\n", heightMap[0 + 0 * (SIZE + 1)]);
	heightMap[SIZE + 0 * (SIZE + 1)] = CORNER;
	printf("heightMap_corner1: %f\n", heightMap[SIZE + 0 * (SIZE + 1)]);
	heightMap[0 + SIZE * (SIZE + 1)] = CORNER;
	printf("heightMap_corner3: %f\n", heightMap[0 + SIZE * (SIZE + 1)]);
	heightMap[SIZE + SIZE * (SIZE + 1)] = CORNER;
	printf("heightMap_corner2: %f\n", heightMap[SIZE + SIZE * (SIZE + 1)]);

	hiprandState* rng;
	/* allocate memory for device */
	hipMalloc(&rng, N * sizeof(hiprandState));
	hipMalloc((void**)&dev_heightMap, N * sizeof(float));
	hipMalloc((void**)&dev_check1, N * sizeof(float));
	hipMalloc((void**)&dev_check2, N * sizeof(float));

	/* memory copy from host to device */
	hipMemcpy(dev_heightMap, heightMap, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_check1, check1, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_check2, check2, N * sizeof(float), hipMemcpyHostToDevice);

	start = clock();
	/* run kernel */
	for (int i = SIZE; i>1; i = i / 2){
		Square_Diamond_100 << <ceil((float)N / 256), 256 >> >(rng, (float*)dev_heightMap, i, dev_check1, dev_check2);
		hipDeviceSynchronize();
	}
	end = clock();

	/* memory copy from device to host*/
	hipMemcpy(heightMap, dev_heightMap, N * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(check1, dev_check1, N * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(check2, dev_check2, N * sizeof(float), hipMemcpyDeviceToHost);

	/* print the output */
	if (print){
		for (int i = 0; i<N; i++){
			printf("%d: x = %d, y = %d; hm = %f\n", i, i % (SIZE + 1), i / (SIZE + 1), heightMap[i]);
		}
	}
	// printf("%f\n", cpu_time_used);
	hipFree(dev_heightMap);
	hipFree(dev_check1);
	hipFree(dev_check2);

	runTime = (double)(end - start) / CLOCKS_PER_SEC;
	printf("Run time for Version_4: %0.20f\n", runTime);
	return EXIT_SUCCESS;
}
